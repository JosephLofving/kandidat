#include "computeTMatrix.h"


template <typename T>
void check(T result, char const *const func, const char *const file,
                     int const line) {
    if (result) {
        fprintf(stderr, "CUDA error at %s:%d code=%d(%s) \"%s\" \n", file, line,
                        static_cast<unsigned int>(result), _cudaGetErrorEnum(result), func);
        exit(EXIT_FAILURE);
    }
}

#define checkCudaErrors(val) check((val), #val, __FILE__, __LINE__)

static const char *_cudaGetErrorEnum(hipError_t error) {
    return hipGetErrorName(error);
}

// Configurable parameters
// Dimension of matrix
// #define N 4
// #define batchSize 2

// Wrapper around malloc
// Clears the allocated memory to 0
// Terminates the program if malloc fails
void* xmalloc(size_t size) {
    void* ptr = malloc(size);
    if (ptr == NULL) {
        printf("> ERROR: malloc for size %zu failed..\n", size);
        exit(EXIT_FAILURE);
    }
    memset(ptr, 0, size);
    return ptr;
}

// void initSetAMatrix(hipDoubleComplex* mat, double factor) {
// 	double toSet[N*N] = {4, 3, 9, 3, 7, 7, 0, 5, 8, 6, 1, 8, 9, 4, 2, 9};
//     for (int i = 0; i < N*N; i++) {
//         mat[i] = make_hipDoubleComplex(toSet[i]*factor, 0); // Scale each element by the factor and set it
//     }
// }

// void initSetBMatrix(hipDoubleComplex* mat, double factor) {
// 	double toSet[N*N] = {53, 36, 16, 50, 74, 59, 33, 67, 78, 69, 47, 61, 191, 148, 79, 166};
//     for (int i = 0; i < N*N; i++) {
//         mat[i] = make_hipDoubleComplex(toSet[i]*factor, 0); // Scale each element by the factor and set it
//     }
// }

// Print column-major matrix
// void printMatrix(hipDoubleComplex* mat, int width, int height) {
//     for (int i = 0; i < height; i++) {
//         for (int j = 0; j < width; j++) {
//             printf("%6.3f ", hipCreal(mat[(j * height) + i]));
//         }
//         printf("\n");
//     }
//     printf("\n");
// }

void computeTMatrixCUBLAS(hipDoubleComplex* d_Tarray,
         			hipDoubleComplex* d_Farray, //h_Farray,
		 			hipDoubleComplex* d_Varray, //h_Varray,
		 			int matLength, int TLabLength) {

    //const int batchSize{ 1 };
    int batchSize = TLabLength;
	// cuBLAS variables
    hipblasStatus_t status;
    hipblasHandle_t handle;

    // Host variables


    // hipDoubleComplex* h_Farray;
    hipDoubleComplex** h_Fptr_array;

    // hipDoubleComplex* h_Varray;
    hipDoubleComplex** h_Vptr_array;

    h_Fptr_array = (hipDoubleComplex**)xmalloc(batchSize * sizeof(hipDoubleComplex*));
    h_Vptr_array = (hipDoubleComplex**)xmalloc(batchSize * sizeof(hipDoubleComplex*));

    // Device variables
    // hipDoubleComplex* d_Farray;
    hipDoubleComplex** d_Fptr_array;

    // hipDoubleComplex* d_Varray;
    hipDoubleComplex** d_Vptr_array;

    int* d_pivotArray;
    int* d_trfInfo;
    int d_trsInfo;

    // Initialize cuBLAS
    status = hipblasCreate(&handle);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf("> ERROR: cuBLAS initialization failed\n");
        //return (EXIT_FAILURE);
    }

    // Allocate memory for host variables
    // h_Farray = (hipDoubleComplex*)xmalloc(batchSize *matLength*matLength* sizeof(hipDoubleComplex));
    // h_Varray = (hipDoubleComplex*)xmalloc(batchSize *matLength*matLength* sizeof(hipDoubleComplex));

    // Allocate memory for device variables

    // checkCudaErrors(hipMalloc((void**)&d_Farray, TLabLength *matLength*matLength* sizeof(hipDoubleComplex)));
    // checkCudaErrors(hipMalloc((void**)&d_Varray, TLabLength *matLength*matLength* sizeof(hipDoubleComplex)));
    checkCudaErrors(
            hipMalloc((void**)&d_pivotArray,matLength* TLabLength * sizeof(int)));
    checkCudaErrors(hipMalloc((void**)&d_trfInfo, TLabLength * sizeof(int)));
    checkCudaErrors(
            hipMalloc((void**)&d_Fptr_array, TLabLength * sizeof(hipDoubleComplex*)));
    checkCudaErrors(
            hipMalloc((void**)&d_Vptr_array, TLabLength * sizeof(hipDoubleComplex*)));

    // for (int i = 0; i < batchSize; i++) {
    //     initSetAMatrix(h_Farray + (i * N*N), (double)(i+1)); // Create matrices scaled by factors 1, 2, ...
    // }

    // printMatrix(h_Farray, N, N);

    // for (int i = 0; i < batchSize; i++) {
    //     initSetBMatrix(h_Varray + (i * N*N), (double)(i+1)); // Create matrices scaled by factors 1, 2, ...
    // }

    // printMatrix(h_Varray, N, N);

    // Copy data to device from host
    // checkCudaErrors(hipMemcpy(d_Farray, h_Farray, TLabLength *matLength*matLength* sizeof(hipDoubleComplex),
                            //    hipMemcpyHostToDevice));
    // checkCudaErrors(hipMemcpy(d_Varray, h_Varray, TLabLength *matLength*matLength* sizeof(hipDoubleComplex),
                            //    hipMemcpyHostToDevice));

    // Create pointer array for matrices
    for (int i = 0; i < TLabLength; i++) h_Fptr_array[i] = d_Farray + (i * matLength * matLength);
    for (int i = 0; i < TLabLength; i++) h_Vptr_array[i] = d_Varray + (i * matLength * matLength);

    // Copy pointer array to device memory
    checkCudaErrors(hipMemcpy(d_Fptr_array, h_Fptr_array,
                               TLabLength * sizeof(hipDoubleComplex*),
							   hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_Vptr_array, h_Vptr_array,
							   TLabLength * sizeof(hipDoubleComplex*),
							   hipMemcpyHostToDevice));

    // Perform LU decomposition
    status = hipblasZgetrfBatched(handle, matLength, d_Fptr_array, matLength, d_pivotArray,
								 d_trfInfo, batchSize);

	// Calculate the T matrix
    status = hipblasZgetrsBatched(handle, HIPBLAS_OP_N, matLength, matLength, d_Fptr_array, matLength,
                                 d_pivotArray, d_Vptr_array, matLength, &d_trsInfo,
								 batchSize);

    // Copy data to host from device
    checkCudaErrors(hipMemcpy(d_Tarray, d_Varray, batchSize *matLength*matLength* sizeof(hipDoubleComplex),
                               hipMemcpyDeviceToDevice));

    // printMatrix(h_Varray, N, N);

    // Free device variables
    checkCudaErrors(hipFree(d_Fptr_array));
    checkCudaErrors(hipFree(d_Vptr_array));
    checkCudaErrors(hipFree(d_trfInfo));
    checkCudaErrors(hipFree(d_pivotArray));
    checkCudaErrors(hipFree(d_Farray));
    checkCudaErrors(hipFree(d_Varray));

    // Free host variables
    // if (h_Farray) free(h_Farray);
    // if (h_Varray) free(h_Varray);

    // Destroy cuBLAS handle
    status = hipblasDestroy(handle);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf("> ERROR: cuBLAS uninitialization failed...\n");
    }
}
