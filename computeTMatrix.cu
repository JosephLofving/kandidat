#include "computeTMatrix.h"


template <typename T>
void check(T result, char const *const func, const char *const file,
                     int const line) {
    if (result) {
        fprintf(stderr, "CUDA error at %s:%d code=%d(%s) \"%s\" \n", file, line,
                        static_cast<unsigned int>(result), _cudaGetErrorEnum(result), func);
        exit(EXIT_FAILURE);
    }
}

#define chkCudaErr(val) check((val), #val, __FILE__, __LINE__)

static const char *_cudaGetErrorEnum(hipError_t error) {
    return hipGetErrorName(error);
}

void computeTMatrixCUBLAS(hipDoubleComplex* T_d,
         			hipDoubleComplex* F_d,
		 			hipDoubleComplex* V_d,
		 			int matLength, int TLabLength) {

    int batchSize = TLabLength;
	// cuBLAS variables
    hipblasStatus_t status;
    hipblasHandle_t handle;

    // Host variables
    hipDoubleComplex** Fptr_array_h;
    hipDoubleComplex** Vptr_array_h;

    Fptr_array_h = (hipDoubleComplex**)malloc(batchSize * sizeof(hipDoubleComplex*));
    Vptr_array_h = (hipDoubleComplex**)malloc(batchSize * sizeof(hipDoubleComplex*));

    // Device variables
    hipDoubleComplex** Fptr_array_d;
    hipDoubleComplex** Vptr_array_d;

    int* pivotArray_d;
    int* trfInfo_d;
    int  trsInfo_d;

    // Initialize cuBLAS
    status = hipblasCreate(&handle);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf("> ERROR: cuBLAS initialization failed\n");
    }

    // Allocate memory for device variables
    chkCudaErr(hipMalloc((void**)&pivotArray_d, matLength * TLabLength * sizeof(int)));
    chkCudaErr(hipMalloc((void**)&trfInfo_d, TLabLength * sizeof(int)));
    chkCudaErr(hipMalloc((void**)&Fptr_array_d, TLabLength * sizeof(hipDoubleComplex*)));
    chkCudaErr(hipMalloc((void**)&Vptr_array_d, TLabLength * sizeof(hipDoubleComplex*)));

    // Create pointer array for matrices
    for (int i = 0; i < TLabLength; i++) Fptr_array_h[i] = F_d + (i * matLength * matLength);
    for (int i = 0; i < TLabLength; i++) Vptr_array_h[i] = V_d + (i * matLength * matLength);

    // Copy pointer array to device memory
    chkCudaErr(hipMemcpy(Fptr_array_d, Fptr_array_h,
                               TLabLength * sizeof(hipDoubleComplex*),
							   hipMemcpyHostToDevice));
    chkCudaErr(hipMemcpy(Vptr_array_d, Vptr_array_h,
							   TLabLength * sizeof(hipDoubleComplex*),
							   hipMemcpyHostToDevice));

    // Perform LU decomposition
    status = hipblasZgetrfBatched(handle, matLength, Fptr_array_d, matLength, pivotArray_d,
								 trfInfo_d, batchSize);

	// Calculate the T matrix
    status = hipblasZgetrsBatched(handle, HIPBLAS_OP_N, matLength, matLength, Fptr_array_d,
                                matLength, pivotArray_d, Vptr_array_d, matLength, &trsInfo_d,
								batchSize);

    // Copy data to host from device
    chkCudaErr(hipMemcpy(T_d, V_d, batchSize*matLength*matLength *
                            sizeof(hipDoubleComplex), hipMemcpyDeviceToDevice));

    // Free device variables
    chkCudaErr(hipFree(Fptr_array_d));
    chkCudaErr(hipFree(Vptr_array_d));
    chkCudaErr(hipFree(trfInfo_d));
    chkCudaErr(hipFree(pivotArray_d));
    chkCudaErr(hipFree(F_d));
    chkCudaErr(hipFree(V_d));

    // Destroy cuBLAS handle
    status = hipblasDestroy(handle);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf("> ERROR: cuBLAS uninitialization failed...\n");
    }
}
