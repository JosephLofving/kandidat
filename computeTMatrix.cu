#include "computeTMatrix.h"


template <typename T>
void check(T result, char const *const func, const char *const file,
                     int const line) {
    if (result) {
        fprintf(stderr, "CUDA error at %s:%d code=%d(%s) \"%s\" \n", file, line,
                        static_cast<unsigned int>(result), _cudaGetErrorEnum(result), func);
        exit(EXIT_FAILURE);
    }
}

#define checkCudaErrors(val) check((val), #val, __FILE__, __LINE__)

static const char *_cudaGetErrorEnum(hipError_t error) {
    return hipGetErrorName(error);
}

// Configurable parameters
// Dimension of matrix
// #define N 4
// #define batchSize 2

// Wrapper around malloc
// Clears the allocated memory to 0
// Terminates the program if malloc fails
void* xmalloc(size_t size) {
    void* ptr = malloc(size);
    if (ptr == NULL) {
        printf("> ERROR: malloc for size %zu failed..\n", size);
        exit(EXIT_FAILURE);
    }
    memset(ptr, 0, size);
    return ptr;
}

void computeTMatrixCUBLAS(hipDoubleComplex* d_Tarray,
         			hipDoubleComplex* d_Farray, //h_Farray,
		 			hipDoubleComplex* d_Varray, //h_Varray,
		 			int matLength, int TLabLength) {

    int batchSize = TLabLength;
	// cuBLAS variables
    hipblasStatus_t status;
    hipblasHandle_t handle;

    // Host variables
    hipDoubleComplex** h_Fptr_array;
    hipDoubleComplex** h_Vptr_array;

    h_Fptr_array = (hipDoubleComplex**)xmalloc(batchSize * sizeof(hipDoubleComplex*));
    h_Vptr_array = (hipDoubleComplex**)xmalloc(batchSize * sizeof(hipDoubleComplex*));

    // Device variables
    hipDoubleComplex** d_Fptr_array;
    hipDoubleComplex** d_Vptr_array;

    int* d_pivotArray;
    int* d_trfInfo;
    int  d_trsInfo;

    // Initialize cuBLAS
    status = hipblasCreate(&handle);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf("> ERROR: cuBLAS initialization failed\n");
        //return (EXIT_FAILURE);
    }

    // Allocate memory for device variables
    checkCudaErrors(hipMalloc((void**)&d_pivotArray,matLength* TLabLength * sizeof(int)));
    checkCudaErrors(hipMalloc((void**)&d_trfInfo, TLabLength * sizeof(int)));
    checkCudaErrors(hipMalloc((void**)&d_Fptr_array, TLabLength * sizeof(hipDoubleComplex*)));
    checkCudaErrors(hipMalloc((void**)&d_Vptr_array, TLabLength * sizeof(hipDoubleComplex*)));

    // Create pointer array for matrices
    for (int i = 0; i < TLabLength; i++) h_Fptr_array[i] = d_Farray + (i * matLength * matLength);
    for (int i = 0; i < TLabLength; i++) h_Vptr_array[i] = d_Varray + (i * matLength * matLength);

    // Copy pointer array to device memory
    checkCudaErrors(hipMemcpy(d_Fptr_array, h_Fptr_array,
                               TLabLength * sizeof(hipDoubleComplex*),
							   hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_Vptr_array, h_Vptr_array,
							   TLabLength * sizeof(hipDoubleComplex*),
							   hipMemcpyHostToDevice));

    // Perform LU decomposition
    status = hipblasZgetrfBatched(handle, matLength, d_Fptr_array, matLength, d_pivotArray,
								 d_trfInfo, batchSize);

	// Calculate the T matrix
    status = hipblasZgetrsBatched(handle, HIPBLAS_OP_N, matLength, matLength, d_Fptr_array,
                                matLength, d_pivotArray, d_Vptr_array, matLength, &d_trsInfo,
								batchSize);

    // Copy data to host from device
    checkCudaErrors(hipMemcpy(d_Tarray, d_Varray, batchSize*matLength*matLength *
                            sizeof(hipDoubleComplex), hipMemcpyDeviceToDevice));

    // Free device variables
    checkCudaErrors(hipFree(d_Fptr_array));
    checkCudaErrors(hipFree(d_Vptr_array));
    checkCudaErrors(hipFree(d_trfInfo));
    checkCudaErrors(hipFree(d_pivotArray));
    checkCudaErrors(hipFree(d_Farray));
    checkCudaErrors(hipFree(d_Varray));

    // Destroy cuBLAS handle
    status = hipblasDestroy(handle);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf("> ERROR: cuBLAS uninitialization failed...\n");
    }
}
