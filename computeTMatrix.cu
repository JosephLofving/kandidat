#include "computeTMatrix.h"


template <typename T>
void check(T result, char const *const func, const char *const file,
                     int const line) {
    if (result) {
        fprintf(stderr, "CUDA error at %s:%d code=%d(%s) \"%s\" \n", file, line,
                        static_cast<unsigned int>(result), _cudaGetErrorEnum(result), func);
        exit(EXIT_FAILURE);
    }
}

#define chkCudaErr(val) check((val), #val, __FILE__, __LINE__)

static const char *_cudaGetErrorEnum(hipError_t error) {
    return hipGetErrorName(error);
}

void computeTMatrixCUBLAS(hipDoubleComplex* d_Tarray,
         			hipDoubleComplex* d_Farray,
		 			hipDoubleComplex* d_Varray,
		 			int matLength, int TLabLength) {

    int batchSize = TLabLength;
	// cuBLAS variables
    hipblasStatus_t status;
    hipblasHandle_t handle;

    // Host variables
    hipDoubleComplex** h_Fptr_array;
    hipDoubleComplex** h_Vptr_array;

    h_Fptr_array = (hipDoubleComplex**)malloc(batchSize * sizeof(hipDoubleComplex*));
    h_Vptr_array = (hipDoubleComplex**)malloc(batchSize * sizeof(hipDoubleComplex*));

    // Device variables
    hipDoubleComplex** d_Fptr_array;
    hipDoubleComplex** d_Vptr_array;

    int* d_pivotArray;
    int* d_trfInfo;
    int  d_trsInfo;

    // Initialize cuBLAS
    status = hipblasCreate(&handle);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf("> ERROR: cuBLAS initialization failed\n");
    }

    // Allocate memory for device variables
    chkCudaErr(hipMalloc((void**)&d_pivotArray, matLength * TLabLength * sizeof(int)));
    chkCudaErr(hipMalloc((void**)&d_trfInfo, TLabLength * sizeof(int)));
    chkCudaErr(hipMalloc((void**)&d_Fptr_array, TLabLength * sizeof(hipDoubleComplex*)));
    chkCudaErr(hipMalloc((void**)&d_Vptr_array, TLabLength * sizeof(hipDoubleComplex*)));

    // Create pointer array for matrices
    for (int i = 0; i < TLabLength; i++) h_Fptr_array[i] = d_Farray + (i * matLength * matLength);
    for (int i = 0; i < TLabLength; i++) h_Vptr_array[i] = d_Varray + (i * matLength * matLength);

    // Copy pointer array to device memory
    chkCudaErr(hipMemcpy(d_Fptr_array, h_Fptr_array,
                               TLabLength * sizeof(hipDoubleComplex*),
							   hipMemcpyHostToDevice));
    chkCudaErr(hipMemcpy(d_Vptr_array, h_Vptr_array,
							   TLabLength * sizeof(hipDoubleComplex*),
							   hipMemcpyHostToDevice));

    // Perform LU decomposition
    status = hipblasZgetrfBatched(handle, matLength, d_Fptr_array, matLength, d_pivotArray,
								 d_trfInfo, batchSize);

	// Calculate the T matrix
    status = hipblasZgetrsBatched(handle, HIPBLAS_OP_N, matLength, matLength, d_Fptr_array,
                                matLength, d_pivotArray, d_Vptr_array, matLength, &d_trsInfo,
								batchSize);

    // Copy data to host from device
    chkCudaErr(hipMemcpy(d_Tarray, d_Varray, batchSize*matLength*matLength *
                            sizeof(hipDoubleComplex), hipMemcpyDeviceToDevice));

    // Free device variables
    chkCudaErr(hipFree(d_Fptr_array));
    chkCudaErr(hipFree(d_Vptr_array));
    chkCudaErr(hipFree(d_trfInfo));
    chkCudaErr(hipFree(d_pivotArray));
    chkCudaErr(hipFree(d_Farray));
    chkCudaErr(hipFree(d_Varray));

    // Destroy cuBLAS handle
    status = hipblasDestroy(handle);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf("> ERROR: cuBLAS uninitialization failed...\n");
    }
}
