#include "computeTMatrix.h"


template <typename T>
void check(T result, char const *const func, const char *const file,
                     int const line) {
    if (result) {
        fprintf(stderr, "CUDA error at %s:%d code=%d(%s) \"%s\" \n", file, line,
                        static_cast<unsigned int>(result), _cudaGetErrorEnum(result), func);
        exit(EXIT_FAILURE);
    }
}

#define chkCudaErr(val) check((val), #val, __FILE__, __LINE__)

static const char *_cudaGetErrorEnum(hipError_t error) {
    return hipGetErrorName(error);
}

void computeTMatrixCUBLAS(hipDoubleComplex* T_d,
         			hipDoubleComplex* F_d,
		 			int matLength, int TLabLength) {

    using microseconds = std::chrono::microseconds;
    auto helaComputeTMatrix_start = std::chrono::high_resolution_clock::now();
    auto bla_start = std::chrono::high_resolution_clock::now();
    auto bla2_start = std::chrono::high_resolution_clock::now();
    //auto bla3_start = std::chrono::high_resolution_clock::now();


	// cuBLAS variables
    hipblasStatus_t status;
    hipblasHandle_t handle;

    // Host variables
    hipDoubleComplex** Fptr_array_h;
    hipDoubleComplex** Tptr_array_h;

    Fptr_array_h = (hipDoubleComplex**)malloc(TLabLength * sizeof(hipDoubleComplex*));
    Tptr_array_h = (hipDoubleComplex**)malloc(TLabLength * sizeof(hipDoubleComplex*));



    // Device variables
    hipDoubleComplex** Fptr_array_d;
    hipDoubleComplex** Tptr_array_d;

    int* pivotArray_d;
    int* trfInfo_d;
    int  trsInfo_d;

    auto bla_stop = std::chrono::high_resolution_clock::now();
    std::cout << "bla:           " << std::chrono::duration_cast<microseconds>(bla_stop - bla_start).count() << "\n";

    // Initialize cuBLAS
    status = hipblasCreate(&handle);
    auto bla2_stop = std::chrono::high_resolution_clock::now();
    std::cout << "bla2:           " << std::chrono::duration_cast<microseconds>(bla2_stop - bla2_start).count() << "\n";
    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf("> ERROR: cuBLAS initialization failed\n");
    }



    // Allocate memory for device variables
    chkCudaErr(hipMalloc((void**)&pivotArray_d, matLength * TLabLength * sizeof(int)));
    chkCudaErr(hipMalloc((void**)&trfInfo_d, TLabLength * sizeof(int)));
    chkCudaErr(hipMalloc((void**)&Fptr_array_d, TLabLength * sizeof(hipDoubleComplex*)));
    chkCudaErr(hipMalloc((void**)&Tptr_array_d, TLabLength * sizeof(hipDoubleComplex*)));




    // Create pointer array for matrices
    for (int i = 0; i < TLabLength; i++) {
        Fptr_array_h[i] = F_d + (i * matLength * matLength);
        Tptr_array_h[i] = T_d + (i * matLength);
    }

    // Copy pointer array to device memory
    chkCudaErr(hipMemcpy(Fptr_array_d, Fptr_array_h,
                               TLabLength * sizeof(hipDoubleComplex*),
							   hipMemcpyHostToDevice));
    chkCudaErr(hipMemcpy(Tptr_array_d, Tptr_array_h,
							   TLabLength * sizeof(hipDoubleComplex*),
							   hipMemcpyHostToDevice));

    // Perform LU decomposition
    status = hipblasZgetrfBatched(handle, matLength, Fptr_array_d, matLength, pivotArray_d,
								 trfInfo_d, TLabLength);

	// Calculate the T matrix
    status = hipblasZgetrsBatched(handle, HIPBLAS_OP_N, matLength, 1, Fptr_array_d,
                                matLength, pivotArray_d, Tptr_array_d, matLength, &trsInfo_d,
								TLabLength);

    // Copy data to host from device
    // chkCudaErr(hipMemcpy(T_d, V_d, TLabLength*matLength*matLength *
    //                         sizeof(hipDoubleComplex), hipMemcpyDeviceToDevice));

    // Free device variables
    chkCudaErr(hipFree(Fptr_array_d));
    chkCudaErr(hipFree(Tptr_array_d));
    chkCudaErr(hipFree(trfInfo_d));
    chkCudaErr(hipFree(pivotArray_d));
    chkCudaErr(hipFree(F_d));
    // chkCudaErr(hipFree(V_d));

    // Destroy cuBLAS handle
    status = hipblasDestroy(handle);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf("> ERROR: cuBLAS uninitialization failed...\n");
    }

    auto helaComputeTMatrix_stop = std::chrono::high_resolution_clock::now();

    //printf("----hela potential:%a\n", std::chrono::duration_cast<microseconds>(helapotential_end - helapotential_start).count());
    //printf("----test: %a\n", std::chrono::duration_cast<microseconds>(test_end - test_start).count());
    std::cout << "hela computeTMatrix: " << std::chrono::duration_cast<microseconds>(helaComputeTMatrix_stop - helaComputeTMatrix_start).count() << "\n";

}
