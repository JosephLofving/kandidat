#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include "mesh.h"
#include <vector>
#include "scattering.h"
#include "potential.h"
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <complex>
#include <fstream>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__
void setupVG(hipDoubleComplex *V, hipDoubleComplex *G0, hipDoubleComplex *VG, int matWidth) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	// printf("Block: %d,%d \tThread: %d,%d\n", blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y);

	// if (row < matWidth && col < matWidth && hipCreal(VG[row + col*matWidth]) != hipCreal(hipCmul(V[row + col*matWidth], G0[col]))) {
	// 	printf("Row: %d  \tcol: %d\t\tGPU: %.2e\tCPU: %.2e\n", row, col, hipCreal(hipCmul(V[row + col*matWidth], G0[col])), hipCreal(VG[row + col*matWidth]));
	// }

	if (row < matWidth && col < matWidth) {
		printf("Block: %d,%d \tThread: %d,%d\n", blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y);
		VG[row + col*matWidth] = hipCmul(V[row + col*matWidth], G0[col]);
	}
}

__global__
void setupVGNonParallell(hipDoubleComplex *V, hipDoubleComplex *G0, hipDoubleComplex *VG, int matrixHeight)
{
	for (int row = 0; row < matrixHeight; row++) {
		for (int col = 0; col < matrixHeight; col++) {
			VG[row+col*matrixHeight] = hipCmul(V[row+col*matrixHeight],G0[col]);
		}
	}
}

int main() {
	const int Nkvadr = 100;
	double scale = 100.0;
	double Tlab = 100.0;

	std::vector<QuantumState> base = setupBase(0, 2, 0, 2);
    std::map<std::string, std::vector<QuantumState> > channels = setupNNChannels(base);
	std::string key = "j:0 s:0 tz:0 pi:1";
	std::vector<QuantumState> channel = channels[key];

	double k0 = getk0(channel, Tlab);

	TwoVectors k_and_w = gaussLegendreInfMesh(Nkvadr, scale);

	std::vector<double> kVect = k_and_w.v1;
	std::vector<double> wVect = k_and_w.v2;

	double* k = &kVect[0];
	double* w = &wVect[0];

	std::vector<std::complex<double>> G0_std = setupG0Vector(channel, kVect, wVect, k0);
	int N = G0_std.size();

	LapackMat V_matrix = potential(channel, kVect, Tlab);
	LapackMat VG_CPU = setupVGKernel(channel, key, V_matrix, kVect, wVect, k0);


	hipDoubleComplex* V_host = new hipDoubleComplex[V_matrix.width*V_matrix.height];

	for (int i = 0; i < N*N; i++) {
		V_host[i] = make_hipDoubleComplex(V_matrix.contents[i].real(), V_matrix.contents[i].imag());
	}


	hipDoubleComplex G0[(Nkvadr+1)];
	for(int i = 0; i < G0_std.size(); i++){
		G0[i] = make_hipDoubleComplex(G0_std[i].real(), G0_std[i].imag());
	}


	hipDoubleComplex* G0_dev;
	hipDoubleComplex* V_dev;
	hipDoubleComplex* VG_dev;

	hipDoubleComplex* VG_host = new hipDoubleComplex[V_matrix.width*V_matrix.height];

	for (int i = 0; i < N*N; i++) {
		VG_host[i] = make_hipDoubleComplex(1.0, 1.0);//make_hipDoubleComplex(VG_CPU.contents[i].real(), VG_CPU.contents[i].imag());
	}

	hipMalloc((void**)&V_dev, N*N*sizeof(hipDoubleComplex));
	hipMalloc((void**)&VG_dev, N*N*sizeof(hipDoubleComplex));
	hipMalloc((void**)&G0_dev, N*sizeof(hipDoubleComplex));

	hipMemcpy(G0_dev, G0, N*sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
	hipMemcpy(V_dev, V_host, N*N*sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
	hipMemcpy(VG_dev, VG_host, N*N*sizeof(hipDoubleComplex), hipMemcpyHostToDevice);

	dim3 threadsPerBlock(N, N);
	dim3 blocksPerGrid(1, 1);
	if (N*N > 512) {
		threadsPerBlock.x = 32;//512;
		threadsPerBlock.y = 32;//512;
		blocksPerGrid.x  = 4;//ceil(double(N)/double(threadsPerBlock.x));
		blocksPerGrid.y  = 4;//ceil(double(N)/double(threadsPerBlock.y));
	}

	printf("%d, %d\n", blocksPerGrid.x, threadsPerBlock.x);

	setupVG <<<blocksPerGrid, threadsPerBlock>>> (V_dev, G0_dev, VG_dev, N);
	//setupVG <<<threadsPerBlock, blocksPerGrid>>> (V_dev, G0_dev, VG_dev, N);
	// setupVGNonParallell <<<1,1>>> (V_dev,G0_dev,VG_dev,N);
	hipDeviceSynchronize();

	// hipDoubleComplex* VG_host= new hipDoubleComplex[V_matrix.width*V_matrix.height];
	// VG_host[5]= make_hipDoubleComplex(1,1);
	hipMemcpy(VG_host, VG_dev, N*N*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);

	//gpuErrchk( hipPeekAtLastError() );

	// std::cout << V_host[0] << std::endl;
	// std::cout << V_host[100] << std::endl;

	hipFree(G0_dev);
	hipFree(V_dev);
	hipFree(VG_dev);

	for (int i = 0; i < N*N; i++) {
		// printf("%d\n", i);
		// if (hipCreal(VG_host[i]) - VG_CPU.contents[i].real() != 0) {
			// printf("Index: %d \t GPU: %f \t CPU: %f\n", i, hipCreal(VG_host[i]), VG_CPU.contents[i].real());
		// }
		printf("GPU: %.2e \t CPU: %.2e\n", hipCreal(VG_host[i]), VG_CPU.contents[i].real());
	}

	return 0;
}
