
#include "tensorAPI.h"


void setElement(hipDoubleComplex* tensor, int row, int column, int slice, int matLength, hipDoubleComplex value){
    tensor[row + column*matLength+slice*matLength*matLength] = value;
}

hipDoubleComplex getElement(hipDoubleComplex* tensor, int row, int column, int slice, int matLength){
    return tensor[row + column*matLength+slice*matLength*matLength];
}