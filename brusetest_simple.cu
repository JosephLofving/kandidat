#include "hip/hip_runtime.h"
#include <cmath>
#include <complex>
#include <iostream>
#include <vector>
#include <fstream>
#include <iomanip>
#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

__global__
void setupVG(double* a)
{
	*a *= 100;
}


int main() {
	int N = 100;

	double kVect = 78;
	double* k = &kVect;
	double* k_dev;
	hipMalloc((void**)&k_dev, N * sizeof(double));
	hipMemcpy(k_dev, k, N * sizeof(double), hipMemcpyHostToDevice);

	std::cout << k << std::endl;
	setupVG << <1, 1 >> > (k_dev);
	hipMemcpy(k, k_dev, N * sizeof(double), hipMemcpyDeviceToHost);

	hipDeviceSynchronize();

	std::cout << k << std::endl;
	hipFree(k_dev);
	std::cout << "hej" << std::endl;

	return 0;
}