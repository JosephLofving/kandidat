#include "hip/hip_runtime.h"
#include <cmath>
#include <complex>
#include <iostream>
#include <vector>
#include <fstream>
#include <iomanip>
#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

__global__
void setupVG(double* a)
{
	*a *= 100;
}


int main() {
	double kVect = 78;
	double* k = &kVect;
	double* k_dev;
	hipMalloc((void**)&k_dev, sizeof(double));
	hipMemcpy(k_dev, k, sizeof(double), hipMemcpyHostToDevice);

	std::cout << *k << std::endl;
	setupVG << <1, 1 >> > (k_dev);
	hipMemcpy(k, k_dev, sizeof(double), hipMemcpyDeviceToHost);

	std::cout << *k << std::endl;
	hipFree(k_dev);

	return 0;
}