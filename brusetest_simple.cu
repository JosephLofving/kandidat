#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__
void setupVG(double* a)
{
	*a = 100;
}


int main() {
	double kVect = 78;
	double* k = &kVect;
	double* k_dev;
	hipMalloc((void**)&k_dev, sizeof(double));
	hipMemcpy(k_dev, k, sizeof(double), hipMemcpyHostToDevice);

	std::cout << *k << std::endl;
	setupVG << <1, 1 >> > (k_dev);
	hipMemcpy(k, k_dev, sizeof(double), hipMemcpyDeviceToHost);

	std::cout << *k << std::endl;
	hipFree(k_dev);

	return 0;
}