#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#define N (32*32)
#define THREADS_PER_BLOCK 32

__global__
void add(int *a, int *b, int *c, n) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n) {
        c[index] = a[index] + b[index];
    }
}

int main() {
    int a[N], b[N], c[N];
    int *dev_a, *dev_b, *dev_c;

    hipMalloc((void**)&dev_c, N*sizeof(int));
    hipMalloc((void**)&dev_b, N*sizeof(int));
    hipMalloc((void**)&dev_a, N*sizeof(int));

    for (int i = 0; i < N; i++) {
        a[i] = -i;
        b[i] = i*i*i;
        // printf("i = %d\n", b[i]);
    }

    hipMemcpy(dev_a, a, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N*sizeof(int), hipMemcpyHostToDevice);

    add <<<N/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(dev_a, dev_b, dev_c, N);

    hipMemcpy(c, dev_c, N*sizeof(int), hipMemcpyDeviceToHost);

    for(int i=0; i < N; i++) {
        printf("%d + %d = %d\n", a[i], b[i], c[i]);
    }

    hipFree(dev_c);
    hipFree(dev_b);
    hipFree(dev_a);

    return 0;
}