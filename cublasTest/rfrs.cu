#include <cblas.h>
#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsolver.h>
#include <iostream>

void printMatrix(double *mat, int width, int height, int matAmt) {
    for (int matNum = 0; matNum < matAmt; matNum++) {
        for (int row = 0; row < height; row++) {
            for (int col = 0; col < width; col++) {
                std::cout << mat[row+col*height] << " ";
            }
            std::cout << ";" << std::endl;
        }
        std::cout << "\n";
    }
}


int main(int argc, char*argv[]) {
    hipblasStatus_t stat;
    hipError_t cudaStatus;
    hipsolverStatus_t cusolverStatus;
    hipblasHandle_t handle;

    int matWidth = 3;
    int matAmt   = 2;

    // Declare arrays on host
    double* A[matAmt];
    double* B[matAmt];

    double[] A1 = {2, -1, 1, 1, 1, 2, 1, -1, 3};
    double[] A2 = {4, -2, 2, 2, 2, 4, 2, -2, 6};

    double[] B1 = {2, 3, -10};
    double[] B2 = {4, 6, -20};

    A[0] = A1; A[1] = A2;
    B[0] = B1; B[1] = B2;

    // double *A, *B; // A - NxN matrix, B1 - auxiliary N-vect, B=A*B - N-vector of RHS, all on the host

    // Declare arrays on device
    double *d_A, *d_B;//, *d_Work; // Coeff matris, RHS, workspace
    int *d_pivot, *d_info;//, Lwork; // Pivots, info, workspace size
    int info_gpu = 0;

    // Prepare memory on host
    A  = (double*)malloc(matAmt*matWidth*matWidth*sizeof(double));
    B  = (double*)malloc(matAmt*matWidth*sizeof(double));

    for (int i = 0; i < matAmt*matWidth*matWidth; i++) {
        A[i] = rand()/(double)RAND_MAX;
    }

    for (int i = 0; i < matAmt; i++) {
        for (int j = 0; j < matWidth; j++) {
            B[i*matWidth+j] = A[i*matWidth*matWidth + j*matWidth];
        }
    }

    std::cout << "A:\n";
    printMatrix(A, matWidth, matWidth, matAmt);
    std::cout << "\nB:\n";
    printMatrix(B, 1, matWidth, matAmt);

    cudaStatus = hipGetDevice(0);

    // Prepare memory on the device
    cudaStatus = hipMalloc((void**)&d_A, matAmt*matWidth*matWidth*sizeof(double));
    cudaStatus = hipMalloc((void**)&d_B,          matAmt*matWidth*sizeof(double));
    cudaStatus = hipMalloc((void**)&d_pivot,      matAmt*matWidth*sizeof(int));
    cudaStatus = hipMalloc((void**)&d_info,                matAmt*sizeof(int));

    cudaStatus = hipMemcpy(d_A, A, matAmt*matWidth*matWidth*sizeof(double), hipMemcpyHostToDevice); // Copy d_A <- A
    cudaStatus = hipMemcpy(d_B, B,          matAmt*matWidth*sizeof(double), hipMemcpyHostToDevice); // Copy d_B <- B

    // BATCHED?
    // cusolverStatus = hipsolverDnSgetrf_bufferSize(handle, matWidth, matWidth, d_A, matWidth, &Lwork); // Compute buffer size and prepare memory

    // cudaStatus = hipMalloc((void**)&d_Work, matAmt*Lwork*sizeof(double));

    stat = hipblasDgetrfBatched(handle, matWidth, &d_A, matWidth, d_pivot, d_info, matAmt);
    stat = hipblasDgetrsBatched(handle, HIPBLAS_OP_N, matWidth, 1, &d_A, matWidth, d_pivot, &d_B, matWidth, d_info, matAmt);

    cudaStatus = hipDeviceSynchronize();

    cudaStatus = hipMemcpy(&info_gpu, d_info, sizeof(int), hipMemcpyDeviceToHost); // d_info -> info_gpu
    cudaStatus = hipMemcpy(B, d_B, matWidth*sizeof(double), hipMemcpyDeviceToHost); // d_B -> B

    printf("\nX:\n");
    printMatrix(B, 1, matWidth, matAmt);
    printf("\n");

    // Free memory
    cudaStatus = hipFree(d_A);
    cudaStatus = hipFree(d_B);
    cudaStatus = hipFree(d_pivot);
    cudaStatus = hipFree(d_info);
    // cudaStatus = hipFree(d_Work);

    free(A); free(B);

    stat = hipblasDestroy(handle);

    cudaStatus = hipDeviceReset();

    return 0;
}