// set cuda api_failures stop
// print *((@global double *)d_B)[0]


#include <cblas.h>
#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsolver.h>
#include <iostream>
#include <csignal>

#define CUBLAS_CALL(x) do { if((x)!=HIPBLAS_STATUS_SUCCESS) { \
      printf("Error at %s:%d\n",__FILE__,__LINE__);\
      exit(-1);}} while(0)

void printMatrix(double *mat, int width, int height, int matAmt) {
    for (int matNum = 0; matNum < matAmt; matNum++) {
        for (int row = 0; row < height; row++) {
            for (int col = 0; col < width; col++) {
                std::cout << mat[row+col*height] << " ";
            }
            std::cout << ";" << std::endl;
        }
    std::cout << std::endl;
    }
}


int main(int argc, char*argv[]) {
    hipblasStatus_t stat;

    hipError_t cudaStatus;
    hipsolverStatus_t cusolverStatus;
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_DEVICE);

    // hipStream_t streamArray;
    // hipStreamCreate(&streamArray);
    // hipblasSetStream(handle, streamArray);

    int matWidth = 3;
    int matAmt   = 2;

    // Declare arrays on host
    // double **h_A[matAmt]; double **h_B[matAmt];


    double h_A[matWidth*matWidth*matAmt] = {2, -1, 1, 1, 1, 2, 1, -1, 3,
                                            4, -2, 2, 2, 2, 4, 2, -2, 6};

    double h_B[matAmt*matWidth] = {2, 3, -10,
                                   4, 6, -20};


    // double *A, *B; // A - NxN matrix, B1 - auxiliary N-vect, B=A*B - N-vector of RHS, all on the host

    // Declare arrays on device
    // double **d_A, **d_B;//, *d_Work; // Coeff matris, RHS, workspace
    double *d_A, *d_B;
    int *d_pivot, *d_info;//, Lwork; // Pivots, info, workspace size
    int info_gpu = 0;

    cudaStatus = hipMalloc((void**)&d_A, matWidth*matWidth*matAmt*sizeof(double));
    cudaStatus = hipMalloc((void**)&d_B,          matWidth*matAmt*sizeof(double));

    double **h_A_pArr = new double *[matAmt];
    double **h_B_pArr = new double *[matAmt];

    double test = 45.0;

    for (int i = 0; i < matAmt; i++) {
        cudaStatus = hipMemcpy(&d_A[i*matWidth*matWidth], &h_A[i*matWidth*matWidth], matWidth*matWidth*sizeof(double), hipMemcpyHostToDevice);
        h_A_pArr[i] = &test;
        // h_A_pArr[i] = &d_A[i*matWidth*matWidth];
    }

    for (int i = 0; i < matAmt; i++) {
        cudaStatus = hipMemcpy(&d_B[i*matWidth], &h_B[i*matWidth], matWidth*sizeof(double), hipMemcpyHostToDevice);
        h_B_pArr[i] = &d_B[i*matWidth];
    }

    double **d_A_pArr;
    double **d_B_pArr;

    cudaStatus = hipMalloc((void**)&d_A_pArr, matAmt*sizeof(double *));
    cudaStatus = hipMalloc((void**)&d_B_pArr, matAmt*sizeof(double *));


    cudaStatus  = hipMemcpy(d_A_pArr, h_A_pArr, matAmt*sizeof(double *), hipMemcpyHostToDevice);
    // cudaStatus  = hipMemcpy(d_A_pArr[1], h_A_pArr[1], matAmt*sizeof(double *), hipMemcpyHostToDevice);
    cudaStatus  = hipMemcpy(d_B_pArr, h_B_pArr, matAmt*sizeof(double *), hipMemcpyHostToDevice);
    // cudaStatus  = hipMemcpy(d_B_pArr[1], h_B_pArr[1], matAmt*sizeof(double *), hipMemcpyHostToDevice);

    // Prepare memory on host
    // A  = (double*)malloc(matAmt*matWidth*matWidth*sizeof(double));
    // B  = (double*)malloc(matAmt*matWidth*sizeof(double));

    // for (int i = 0; i < matAmt*matWidth*matWidth; i++) {
    //     A[i] = rand()/(double)RAND_MAX;
    // }

    // for (int i = 0; i < matAmt; i++) {
    //     for (int j = 0; j < matWidth; j++) {
    //         B[i*matWidth+j] = A[i*matWidth*matWidth + j*matWidth];
    //     }
    // }

    // std::cout << "A:\n";
    // printMatrix(h_A1, matWidth, matWidth);
    // std::cout << std::endl;
    // printMatrix(h_A2, matWidth, matWidth);
    // std::cout << "\nB:\n";
    // printMatrix(h_B1, 1, matWidth);
    // std::cout << std::endl;
    // printMatrix(h_B2, 1, matWidth);

                //cudaStatus = hipGetDevice(0);

    // Prepare memory on the device
    // cudaStatus = hipMalloc((void**)&d_A1, matWidth*matWidth*sizeof(double));
    // cudaStatus = hipMalloc((void**)&d_A2, matWidth*matWidth*sizeof(double));
    // cudaStatus = hipMalloc((void**)&d_B1,          matWidth*sizeof(double));
    // cudaStatus = hipMalloc((void**)&d_B2,          matWidth*sizeof(double));
    cudaStatus = hipMalloc((void**)&d_pivot,   matAmt*matWidth*sizeof(int));
    cudaStatus = hipMalloc((void**)&d_info,                    sizeof(int));

    // cudaStatus = hipMemcpy(d_A1, h_A1, matWidth*matWidth*sizeof(double), hipMemcpyHostToDevice); // Copy d_A <- A
    // cudaStatus = hipMemcpy(d_A2, h_A2, matWidth*matWidth*sizeof(double), hipMemcpyHostToDevice); // Copy d_A <- A
    // cudaStatus = hipMemcpy(d_B1, h_B1,          matWidth*sizeof(double), hipMemcpyHostToDevice); // Copy d_B <- B
    // cudaStatus = hipMemcpy(d_B2, h_B2,          matWidth*sizeof(double), hipMemcpyHostToDevice); // Copy d_B <- B

    // double **h_A = new double *[matAmt];
    // double **h_B = new double *[matAmt];

    // double *d_A; double *d_B;

    // cudaStatus = hipMalloc((void**)&d_A, matWidth*matWidth)

    // double *h_A[matAmt]; double *h_B[matAmt];
    // double *d_A[matAmt]; double *d_B[matAmt];

    // h_A[0] = d_A1; h_A[1] = d_A2;
    // h_B[0] = d_B1; h_B[1] = d_B2;

    // cudaStatus = hipMalloc((void**)&d_A, matAmt*sizeof(double*));
    // cudaStatus = hipMalloc((void**)&d_B, matAmt*sizeof(double*));

    // cudaStatus = hipMemcpy(d_A, h_A, matAmt*sizeof(double*), hipMemcpyHostToDevice);
    // cudaStatus = hipMemcpy(d_B, h_B, matAmt*sizeof(double*), hipMemcpyHostToDevice);

    // BATCHED?
    // cusolverStatus = hipsolverDnSgetrf_bufferSize(handle, matWidth, matWidth, d_A, matWidth, &Lwork); // Compute buffer size and prepare memory

    // cudaStatus = hipMalloc((void**)&d_Work, matAmt*Lwork*sizeof(double));

    // int *h_info;

    CUBLAS_CALL( hipblasDgetrfBatched(handle, matWidth, d_A_pArr, matWidth, d_pivot, d_info, matAmt) ); // h_A -> d_A?
    CUBLAS_CALL( hipblasDgetrsBatched(handle, HIPBLAS_OP_N, matWidth, 1, d_A_pArr, matWidth, d_pivot, d_B_pArr, matWidth, d_info, matAmt) );

    // stat = hipblasDgetrfBatched(handle, matWidth, h_A, matWidth, d_pivot, d_info, matAmt); // h_A -> d_A?
    // stat = hipblasDgetrsBatched(handle, HIPBLAS_OP_N, matWidth, 1, h_A, matWidth, d_pivot, h_B, matWidth, d_info, matAmt);

    cudaStatus = hipDeviceSynchronize();

    // cudaStatus = hipMemcpy(&info_gpu, d_info, sizeof(int), hipMemcpyDeviceToHost); // d_info -> info_gpu
    cudaStatus = hipMemcpy(h_B, d_B, matWidth*sizeof(double), hipMemcpyDeviceToHost); // d_B -> B

    printMatrix(h_B, 1, matWidth, 2);

    // printf("\nX:\n");
    // printMatrix(h_B1, 1, matWidth);
    // printf("\n");
    // printMatrix(h_B2, 1, matWidth);
    // printf("\n");

                // 0x555559d78748 <_Z14getrf_semiwarpIddLi2ELi3ELb1EEviPKPT_iPiS4_i+520>:  0x00010c0c

    // Free memory
    cudaStatus = hipFree(d_A);
    cudaStatus = hipFree(d_A_pArr);
    cudaStatus = hipFree(d_B);
    cudaStatus = hipFree(d_B_pArr);
    cudaStatus = hipFree(d_pivot);
    cudaStatus = hipFree(d_info);
    // cudaStatus = hipFree(d_Work);

    free(h_A); free(h_B);

    stat = hipblasDestroy(handle);

    cudaStatus = hipDeviceReset();

    return 0;
}