#include <stdio.h>
#include <hip/hip_runtime.h>

#include <hiprand/hiprand_kernel.h>

#define N (1290)

__global__
void add(int *a, int *b, int *c) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    while (tid < N) {
        c[tid] = a[tid] + b[tid];
        tid += blockDim.x*gridDim.x;
    }
}

int main() {
    int a[N], b[N], c[N];
    int *dev_a, *dev_b, *dev_c;

    hipMalloc((void**)&dev_c, N*sizeof(int));
    hipMalloc((void**)&dev_b, N*sizeof(int));
    hipMalloc((void**)&dev_a, N*sizeof(int));

    for (int i = 0; i < N; i++) {
        a[i] = -i;
        b[i] = i*i*i;
        // printf("i = %d\n", b[i]);
    }

    hipMemcpy(dev_a, a, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N*sizeof(int), hipMemcpyHostToDevice);

    add<<<128, 128>>>(dev_a, dev_b, dev_c);

    hipMemcpy(c, dev_c, N*sizeof(int), hipMemcpyDeviceToHost);

    for(int i=0; i < N; i++) {
        printf("%d + %d = %d\n", a[i], b[i], c[i]);
    }

    hipFree(dev_c);
    hipFree(dev_b);
    hipFree(dev_a);

    return 0;
}