int main() {
	hipMalloc((void**)&V_dev, N * N * sizeof(double));
	hipMemcpy(G0_dev, G0, N * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(V_host, VG_dev, N * N * sizeof(double), hipMemcpyDeviceToHost);
	hipFree(G0_dev);
}