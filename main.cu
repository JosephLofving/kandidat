#include "hip/hip_runtime.h"
#include "mesh.h"
#include "scattering.h"
#include "potential.h"
#include <fstream>
#include <iomanip>

#include <hip/hip_complex.h>

__global__
void setupVG(double *k, double *w, hipDoubleComplex *V, double k0, hipDoubleComplex *G0, hipDoubleComplex *VG, int matrixHeight)
{
	for (int row = 0; row < matrixHeight; row++) {
		for (int column = 0; column < matrixHeight; column++) {
			VG[row+column*matrixHeight] = hipCmul(V[row+column*matrixHeight],G0[column]);
		}
	}
}

int main() {
	std::vector<QuantumState> base = setupBase(0, 2, 0, 2);
    std::map<std::string, std::vector<QuantumState> > channels = setupNNChannels(base);
	printChannels(channels);

	int NKvadratur = 100;
	double scale = 100;


	std::string key = "j:0 s:0 tz:0 pi:1";
	std::vector<QuantumState> channel = channels[key];
	if (channel.size()==0) {
		std::cout << "Invalid key";
		abort();
	}
	printStates(channel);

	double Tlab = 100.0;

//------------------------------------------------------------------
//-------------------------- FOR GPU --------------------------------
//------------------------------------------------------------------

	double k0 = getk0(channel, Tlab);

	TwoVectors k_and_w = gaussLegendreInfMesh(NKvadratur, scale);

	std::vector<double> kVect = k_and_w.v1;
	double* k = &kVect[0];
	std::vector<double> wVect = k_and_w.v2;
	double* w = &wVect[0];

std::vector<std::complex<double>> G0_std = setupG0Vector(channel, kVect, wVect, k0);

	int N = kVect.size();

	LapackMat V_matrix = potential(channel, kVect, Tlab);
	hipDoubleComplex *V = new hipDoubleComplex[N*N];

	for (int i = 0; i < V_matrix.height*V_matrix.width; i++) {
		V[i] = make_hipDoubleComplex(V_matrix.contents[i].real(), V_matrix.contents[i].imag());
	}

	hipDoubleComplex G0[N];

	for(int i = 0; i < G0_std.size(); i++){
		G0[i] = make_hipDoubleComplex(G0_std[i].real(), G0_std[i].imag());
	}

	double *k_dev;// = (double*) malloc(N*sizeof(double));
	double *w_dev;// = (double*) malloc(N*sizeof(double));
	hipDoubleComplex *V_dev;
	hipDoubleComplex *G0_dev;
	hipDoubleComplex *VG_dev;

	hipMalloc((void**)&k_dev, N*sizeof(double));
	hipMalloc((void**)&w_dev, N*sizeof(double));
	hipMalloc((void**)&V_dev, N*N*sizeof(hipDoubleComplex));
	hipMalloc((void**)&G0_dev, N*sizeof(hipDoubleComplex));
	hipMalloc((void**)&VG_dev, N*N*sizeof(hipDoubleComplex));

	hipMemcpy(&k_dev, &k, N*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(&w_dev, &w, N*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(&V_dev, &V, N*N*sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
	hipMemcpy(&G0_dev, &G0, N*sizeof(hipDoubleComplex), hipMemcpyHostToDevice);

	setupVG<<<1, 1>>>(k_dev, w_dev, V_dev, k0, G0_dev, VG_dev, N);

	hipDeviceSynchronize();

	hipDoubleComplex VG[N*N];
	hipMemcpy(&VG, &VG_dev, N*N*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);

	std::cout << hipCreal(VG[0]);

	hipFree(VG_dev);
	hipFree(V_dev);
	hipFree(G0_dev);
	hipFree(k_dev);
	hipFree(w_dev);

	return 0;
}