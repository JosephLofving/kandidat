#include "hip/hip_runtime.h"
#include "mesh.h"
#include "scattering.h"
#include "potential.h"
#include <fstream>
#include <iomanip>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_complex.h>



/* Kvar att g�ra som jag kan komma p� just nu (innan vi �r redo att testa t.ex. setupVGKernel)
 * - Fixa block/threads osv i scatteringfilerna samt i kernel calls (gl�m inte getk0)
 * - Fixa computePhaseShifts och BlattToStapp s� att de g�r att k�ra parallellt (de �r typ or�rda nu)
 */







/**
	Gets the reduced mass by checking the isospin channel, which determines the type of NN scattering
	@param channel:	Scattering channel
	@return			Reduced mass
*/
double getReducedMass(std::vector<QuantumState> channel) {
	double mu = 0;
	int tzChannel = channel[0].state["tz"];
	if (tzChannel == -1)	 // Proton-proton scattering
		mu = constants::protonMass / 2;
	else if (tzChannel == 0) // Proton-neutron scattering
		mu = constants::nucleonReducedMass;
	else if (tzChannel == 1) // Neutron-neutron scattering
		mu = constants::neutronMass / 2;

	return mu;
}


/**
	Checks if the state is coupled or not.
	@param channel: Scattering channel
	@return			True if coupled, false if not
*/
bool isCoupled(std::vector<QuantumState> channel) {
	/* If there is only one channel the state is uncoupled, otherwise there are four channels and the state is coupled. */
	return !(channel.size() == 1);
}


__global__
void getk0(double* k0, double* TLab, int TLabLength, int tzChannel) {
	double k0Squared = 0;
	for (int i = 0; i < TLabLength; i++) {
		/* Proton-proton scattering */
		if (tzChannel == -1) {
			k0Squared = constants::protonMass * TLab[i] / 2;
		}
		/* Proton-neutron scattering */
		else if (tzChannel == 0) {
			k0Squared = pow(constants::neutronMass, 2) * TLab[i] * (TLab[i]
				+ 2 * constants::protonMass) / ((pow(constants::protonMass
					+ constants::neutronMass, 2) + 2 * TLab[i] * constants::neutronMass));
		}
		/* Neutron-neutron scattering */
		else if (tzChannel == 1) {
			k0Squared = constants::neutronMass * TLab[i] / 2;
		}

		k0[i] = sqrtf(k0Squared); // Does not handle case where tz is NOT -1, 0 or 1 (should be handled earlier?)
	}
}






int main() {
	std::vector<QuantumState> base = setupBase(0, 2, 0, 2);
	std::map<std::string, std::vector<QuantumState>> channels = setupNNChannels(base);

	std::string key = "j:0 s:0 tz:0 pi:1";
	std::vector<QuantumState> channel = channels[key];
	if (channel.size() == 0) {
		std::cout << "Invalid key";
		abort();
	}
	int tzChannel = channel[0].state["tz"];


	/* Set up Gauss-Legendre quadrature */
	int quadratureN = 100;
	double scale = 100;
	kAndWPtrs kAndW = gaussLegendreInfMesh(quadratureN, scale);
	double* k_h = kAndW.k;
	double* w_h = kAndW.w;

	/* Determine matrix and vector sizes */
	int matSize;
	int phasesSize;
	bool coupled = isCoupled(channel);
	if (coupled) {
		matSize = 2 * (quadratureN + 1);	// a trick to facilitate calculations (see scattering.cu)
		phasesSize = 3; // two phase shifts and one mixing angle in coupled case
	}
	else {
		matSize = quadratureN + 1;
		phasesSize = 1; // only one phase shift in uncoupled case
	}

	/* Prepare generation of TLab [Mev] */
	double TLabMin = 100;
	double TLabMax = 100;
	double TLabIncr = 1;
	int TLabLength = static_cast<int>( (TLabMax - TLabMin) / TLabIncr + 1);


	/* Allocate host memory */
	double* TLab_h = new double[TLabLength];
	double* k0_h = new double[TLabLength];
	hipDoubleComplex* V_h = new hipDoubleComplex[matSize * matSize * TLabLength];
	hipDoubleComplex* T_h = new hipDoubleComplex[matSize * matSize];
	hipDoubleComplex* G0_h = new hipDoubleComplex[matSize];
	hipDoubleComplex* VG_h = new hipDoubleComplex[matSize * matSize];
	hipDoubleComplex* F_h = new hipDoubleComplex[matSize * matSize];
	hipDoubleComplex* phases_h = new hipDoubleComplex[phasesSize];
	

	/* Allocate device memory */
	double* TLab_d;
	double* k0_d;
	double* k_d;
	double* w_d;
	hipDoubleComplex* V_d;
	hipDoubleComplex* T_d;
	hipDoubleComplex* G0_d;
	hipDoubleComplex* VG_d;
	hipDoubleComplex* F_d;
	hipDoubleComplex* phases_d;
	hipMalloc((void**)&TLab_d, TLabLength * sizeof(double));
	hipMalloc((void**)&k0_d, TLabLength * sizeof(double));
	hipMalloc((void**)&k_d, quadratureN * sizeof(double));
	hipMalloc((void**)&w_d, quadratureN * sizeof(double));
	hipMalloc((void**)&V_d, matSize * matSize * TLabLength * sizeof(hipDoubleComplex));
	hipMalloc((void**)&G0_d, matSize * matSize * sizeof(hipDoubleComplex));
	hipMalloc((void**)&VG_d, matSize * matSize * sizeof(hipDoubleComplex));
	hipMalloc((void**)&F_d, matSize * matSize * sizeof(hipDoubleComplex));
	hipMalloc((void**)&T_d, matSize * matSize * sizeof(hipDoubleComplex));
	hipMalloc((void**)&phases_d, phasesSize * sizeof(hipDoubleComplex));


	/* Copy host variables to device variables */
	hipMemcpy(TLab_d, TLab_h, TLabLength * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(k0_d, k0_h, TLabLength * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(k_d, k_h, quadratureN * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(w_d, w_h, quadratureN * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(G0_d, G0_h, matSize * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
	hipMemcpy(VG_d, VG_h, matSize * matSize * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
	hipMemcpy(F_d, F_h, matSize * matSize * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
	hipMemcpy(T_d, T_h, matSize * matSize * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
	hipMemcpy(phases_d, phases_h, phasesSize * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);


	/* Generate different experimental kinetic energies [MeV]*/
	for (int i = 0; i < TLabLength; i++) {
		TLab_h[i] = i * TLabIncr;
	}

	getk0<<<1, 1 >>>(k0_d, TLab_d, TLabLength, tzChannel);

	hipMemcpy(k0_h, k0_d, TLabLength * sizeof(double), hipMemcpyDeviceToHost);

	/* Create the potential matrix on the CPU */
	for (int i = 0; i < TLabLength; i++) {
		V_h[i] = potential(channel, k_h, TLab_h[i], k0_h[i], quadratureN)[i];
	}

	double mu = getReducedMass(channel);

	dim3 threadsPerBlock(matSize, matSize);
	dim3 blocksPerGrid(1, 1);
	if (matSize * matSize > 512) {
		threadsPerBlock.x = 512;
		threadsPerBlock.y = 512;
		blocksPerGrid.x = ceil(double(matSize) / double(threadsPerBlock.x));
		blocksPerGrid.y = ceil(double(matSize) / double(threadsPerBlock.y));
	}

	/* Call kernels on GPU */
	computeTMatrix <<<threadsPerBlock, blocksPerGrid>>> (T_d, V_d, G0_d, VG_d, F_d, k_d, w_d, k0_d, quadratureN, matSize, mu, coupled);
	//computePhaseShifts <<<threadsPerBlock, blocksPerGrid>>> (phases_h, mu, coupled, k0_d, T_d, quadratureN);
	
	hipDeviceSynchronize();

	/* Copy (relevant) device variables to host variables */
	hipMemcpy(T_h, T_d, matSize * matSize * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
	hipMemcpy(phases_h, phases_d, phasesSize * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);

	hipMemcpy(VG_h, VG_d, matSize * matSize * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);

	for (int i = 0; i < matSize * matSize; i += 5) {
		std::cout << hipCreal(VG_h[i]) << std::endl;
	}
	//-------------------------------------------
	// perhaps some printing of T or phases here
	//-------------------------------------------

	/* Free all the allocated memory */ 
	delete[] TLab_h;
	delete[] k0_h;
	delete[] V_h;
	delete[] G0_h;
	delete[] VG_h;
	delete[] F_h;
	delete[] T_h;
	delete[] phases_h;
	hipFree(TLab_d);
	hipFree(k0_d);
	hipFree(V_d);
	hipFree(G0_d);
	hipFree(VG_d);
	hipFree(F_d);
	hipFree(T_d);
	hipFree(phases_d);

	return 0;
}