#include "hip/hip_runtime.h"
#include "mesh.h"
#include "scattering.h"
#include "potential.h"
#include <fstream>
#include <iomanip>

#include <hip/hip_complex.h>

// Borde vara V^T * G0 iställer
__global__
void setupVG(double *k, double *w, hipDoubleComplex *V, double k0, hipDoubleComplex *G0, hipDoubleComplex *VG, int matrixHeight)
{
	for (int row = 0; row < matrixHeight; row++) {
		for (int column = 0; column < matrixHeight; column++) {
			VG[row+column*matrixHeight] = hipCmul(V[row+column*matrixHeight],G0[column]);
		}
	}
}

int main() {
	/*
	std::ofstream myfile;
    myfile.open ("data.csv");

	myfile << "Real av fasskift";
	myfile << ",";
	myfile << "N";
	myfile << "\n";
	*/

	std::vector<QuantumState> base = setupBase(0, 2, 0, 2);
    std::map<std::string, std::vector<QuantumState> > channels = setupNNChannels(base);
	printChannels(channels);

	int NKvadratur = 100;
	double scale = 100;


	std::string key = "j:0 s:0 tz:0 pi:1";
	std::vector<QuantumState> channel = channels[key];
	if (channel.size()==0) {
		std::cout << "Invalid key";
		abort();
	}
	printStates(channel);

	double Tlab = 100.0;

//------------------------------------------------------------------
//-------------------------- FOR GPU --------------------------------
//------------------------------------------------------------------

	std::vector<double> k;
	std::vector<double> w;



	double k0 = getk0(channel, Tlab);



	// Allocate Unified Memory, i.e. let the following objects be accessible
	// from both GPU and CPU


	// Initialize k, w, V_matrix on CPU
	TwoVectors k_and_w{ gaussLegendreInfMesh(NKvadratur, scale) };
	k = k_and_w.v1;
	w = k_and_w.v2;

std::vector<std::complex<double>> G0_std = setupG0Vector(channel, k, w, k0);

	int N = k.size();

	double *k_dev;
	double *w_dev;
	hipDoubleComplex *V_dev;
	hipDoubleComplex *G0_dev;
	hipDoubleComplex *VG_dev;

	LapackMat V_matrix = potential(channel, k, Tlab);
	hipDoubleComplex *V = new hipDoubleComplex[N*N];

	for (int i = 0; i < V_matrix.height*V_matrix.width; i++) {
		V[i] = make_hipDoubleComplex(V_matrix.contents[i].real(), V_matrix.contents[i].imag());
	}

	hipDoubleComplex G0[N];

	for(int i = 0; i < G0_std.size(); i++){
		G0[i] = make_hipDoubleComplex(G0_std[i].real(), G0_std[i].imag());
	}

	hipMalloc((void**)&k_dev, N*sizeof(double));
	hipMalloc((void**)&w_dev, N*sizeof(double));
	hipMalloc((void**)&V_dev, N*N*sizeof(hipDoubleComplex));
	hipMalloc((void**)&G0_dev, N*sizeof(hipDoubleComplex));
	hipMalloc((void**)&VG_dev, N*N*sizeof(hipDoubleComplex));

	hipMemcpy(&k_dev, &k, N*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(&w_dev, &w, N*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(&V_dev, &V, N*N*sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
	hipMemcpy(&G0_dev, &G0, N*sizeof(hipDoubleComplex), hipMemcpyHostToDevice);

	setupVG<<<1, 1>>>(k_dev, w_dev, V_dev, k0, G0_dev, VG_dev, N);

	hipDoubleComplex VG[N*N];

	hipMemcpy(&VG, &VG_dev, N*N*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);

	// Compute the phase shifts for many different T matrices
	//std::vector<std::complex<double>> phase = computePhaseShifts<<<1,1>>>(channel, key, k0, T);

	// Wait for GPU to finish before accessing on host
	hipDeviceSynchronize();

	std::cout << hipCreal(VG[0]);


	// Free memory from Unified Memory
	// hipFree(VG_dev);
	// hipFree(V_dev);
	// hipFree(G0_dev);
	// hipFree(k_dev);
	// hipFree(w_dev);

	return 0;
}