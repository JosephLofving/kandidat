#include "hip/hip_runtime.h"
#include "scattering.h"

__device__
void cudaSetElement(hipDoubleComplex* tensor, int row, int column, int slice, int matLength, hipDoubleComplex value){
    tensor[row + column*matLength+slice*matLength*matLength] = value;
}

__device__
hipDoubleComplex cudaGetElement(hipDoubleComplex* tensor, int row, int column, int slice, int matLength){
    return tensor[row + column*matLength+slice*matLength*matLength];
}

__device__
hipDoubleComplex operator+(hipDoubleComplex A, hipDoubleComplex B) {
	hipDoubleComplex result = make_hipDoubleComplex(hipCreal(A)+hipCreal(B), hipCimag(A)+hipCimag(B));
	return result;
}

__device__
hipDoubleComplex operator-(hipDoubleComplex A, hipDoubleComplex B) {
	hipDoubleComplex result = make_hipDoubleComplex(hipCreal(A) - hipCreal(B), hipCimag(A) - hipCimag(B));
	return result;
}

__device__
hipDoubleComplex operator-(double a, hipDoubleComplex A) {
	hipDoubleComplex result = hipCsub(make_hipDoubleComplex(a, 0), A);
	return result;
}

__device__
hipDoubleComplex operator-(hipDoubleComplex A, double a) {
	hipDoubleComplex result = hipCsub(A, make_hipDoubleComplex(a, 0));
	return result;
}

__device__
hipDoubleComplex operator*(double scalar, hipDoubleComplex A) {
	hipDoubleComplex result = make_hipDoubleComplex(scalar * hipCreal(A), scalar * hipCimag(A));
	return result;
}

__device__
hipDoubleComplex operator*(hipDoubleComplex A, double scalar) {
	return scalar * A;
}

__device__
hipDoubleComplex operator*(hipDoubleComplex A, hipDoubleComplex B) {
	hipDoubleComplex realProd = hipCreal(A) * B;
	hipDoubleComplex imagProd = make_hipDoubleComplex(-hipCimag(A) * hipCimag(B), hipCimag(A)*hipCreal(B));
	hipDoubleComplex result = hipCadd(realProd, imagProd);
	return result;
}

__device__
hipDoubleComplex operator/(hipDoubleComplex A, hipDoubleComplex B) {
	return hipCdiv(A, B);
}

__device__
hipDoubleComplex operator/(hipDoubleComplex A, double a) {
	return hipCdiv(A, make_hipDoubleComplex(a, 0));
}

__device__
hipDoubleComplex operator/(double a, hipDoubleComplex A) {
	return hipCdiv(make_hipDoubleComplex(a, 0), A);
}

__device__
hipDoubleComplex logCudaComplex(hipDoubleComplex argument) {
	double x = hipCreal(argument);
	double y = hipCimag(argument);
	double real = logf(sqrtf(x * x + y * y));
	double imag = atan2f(y, x);
	hipDoubleComplex result = make_hipDoubleComplex(real, imag);
	return result;
}

__device__
double signCuda(double argument) {
	if (argument > 0) return 1;
	else if (argument == 0) return 0;
	else return -1;
}

__device__
hipDoubleComplex sqrtCudaComplex(hipDoubleComplex argument) {
	double x = hipCreal(argument);
	double y = hipCimag(argument);
	double real = sqrtf((sqrtf(x * x + y * y) + x) / 2);
	double imag = signCuda(y) * sqrtf((sqrtf(x * x + y * y) - x) / 2);
	return make_hipDoubleComplex(real, imag);
}

__device__
hipDoubleComplex atanCudaComplex(hipDoubleComplex argument) {
	hipDoubleComplex numerator = hipCadd(make_hipDoubleComplex(1, 0), hipCmul(make_hipDoubleComplex(0, 1), argument));
	hipDoubleComplex denominator = hipCsub(make_hipDoubleComplex(1, 0), hipCmul(make_hipDoubleComplex(0, 1), argument));
	hipDoubleComplex logOfStuff = logCudaComplex(hipCdiv(numerator, denominator));
	hipDoubleComplex result = hipCmul(make_hipDoubleComplex(0, -0.5), logOfStuff);
	return result;
}

__device__
hipDoubleComplex asinCudaComplex(hipDoubleComplex argument) {
	const hipDoubleComplex I = make_hipDoubleComplex(0.0, 1.0);
	return I * logCudaComplex(sqrtCudaComplex(1 - argument * argument) - I * argument);
}

__device__
hipDoubleComplex expCudaComplex(hipDoubleComplex argument) {
	double x = hipCreal(argument);
	double y = hipCimag(argument);
	hipDoubleComplex trig = make_hipDoubleComplex(cosf(y), sinf(y));
	hipDoubleComplex result = make_hipDoubleComplex(expf(x), 0) * trig;
	return result;
}

__device__
hipDoubleComplex sinCudaComplex(hipDoubleComplex argument) {
	const hipDoubleComplex I = make_hipDoubleComplex(0.0, 1.0);
	return (expCudaComplex(I * argument) - expCudaComplex(-1.0 * I * argument)) / 2;
}

__device__
hipDoubleComplex tanCudaComplex(hipDoubleComplex argument) {
	const hipDoubleComplex I = make_hipDoubleComplex(0.0, 1.0);
	hipDoubleComplex numerator = I * (expCudaComplex(-1.0 * I * argument) - expCudaComplex(I * argument));
	hipDoubleComplex denominator = expCudaComplex(-1.0 * I * argument) + expCudaComplex(I * argument);
	return numerator / denominator;
}

__global__
void setupG0Vector(hipDoubleComplex* G0,
	double* k,
	double* w,
	double* k0,
	double* sum,
	int quadratureN,
	int matLength,
	int TLabLength,
	double mu,
	bool coupled) {

	int column = blockIdx.x * blockDim.x + threadIdx.x;
	int slice = blockIdx.z * blockDim.z + threadIdx.z;

	double twoMu = (2.0 * mu);
	double twoOverPi = (2.0 / constants::pi);

	if (column < quadratureN && slice < TLabLength) {
		G0[column + slice * matLength] = make_hipDoubleComplex(twoOverPi * twoMu * k[column] * k[column] * w[column] / (k0[slice] * k0[slice] - k[column] * k[column]), 0);

		/* If coupled, append G0 to itself to facilitate calculations.
		 * This means the second half of G0 is a copy of the first. */
		if (coupled) {
			G0[quadratureN + 1 + column + slice * matLength] = G0[column + slice * matLength];
		}

		/* Assign the last element of G0 */
		G0[quadratureN + slice * matLength] = make_hipDoubleComplex(-twoOverPi * twoMu * k0[slice] * k0[slice] * sum[slice], -twoMu * k0[slice]);
		if (coupled) {
			G0[2 * (quadratureN + 1) - 1 + slice * matLength] = G0[quadratureN + slice * matLength];
		}
	}
}

/**
	Multiplies the potential matrix elements with the G0 vector elements.

	@param channel: Scattering channel
	@param key:		Channel name
	@param V:		Potential matrix
	@param k:		Quadrature points
	@param w:		Weights for quadrature points
	@param k0:		On-shell-point
	@return			VG kernel
*/
__global__
void setupVGKernel(hipDoubleComplex* VG,
	hipDoubleComplex* V,
	hipDoubleComplex* G0,
	hipDoubleComplex* F,
	double* k,
	double* w,
	double* k0,
	int quadratureN,
	int matLength,
	int TLabLength,
	double mu,
	bool coupled) {

	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int column = blockIdx.x * blockDim.x + threadIdx.x;
	int slice = blockIdx.z * blockDim.z + threadIdx.z;

	if (row < matLength && column < matLength && slice < TLabLength) {
		VG[row + column * matLength + slice * matLength * matLength] = hipCmul(V[row + column * matLength + slice * matLength * matLength], G0[column + slice * matLength]);

		if (row == column) {
			F[row + row * matLength + slice * matLength * matLength] = hipCadd(make_hipDoubleComplex(1, 0), hipCmul(make_hipDoubleComplex(-1, 0), VG[row + row * matLength + slice * matLength * matLength])); // Diagonal element
		}
		else {
			F[row + column * matLength + slice * matLength * matLength] = hipCmul(make_hipDoubleComplex(-1, 0), VG[row + column * matLength + slice * matLength * matLength]);
		}

	}

}



	//for (int row = 0; row < matLength; row++) {
	//	for (int column = 0; column < matLength; column++) {
	//		/* Create VG by using VG[i,j] = V[i,j] * G[j] */
	//		VG[row + column * matLength] = hipCmul(V[row + column * matLength], G0[column]);

	//		/* At the same time, create F = delta_ij - VG_ij for computeTMatrix*/
	//		if (row != column) {
	//			F[row + column * matLength] = hipCmul(make_hipDoubleComplex(-1, 0), VG[row + column * matLength]);
	//		}
	//	}
	//	F[row + row * matLength] = hipCadd(make_hipDoubleComplex(1, 0), hipCmul(make_hipDoubleComplex(-1, 0), VG[row + row * matLength])); // Diagonal element
	//}
//}




/**
	Computes the T-matrix from the equation [F][T] = [V]

	@param channel: Scattering channel
	@param key:		Channel name
	@param V:		Potential matrix
	@param k:		Quadrature points
	@param w:		Weights for quadrature points
	@param k0:		On-shell-point
	@return			T matrix
*/


/* TODO: Explain theory for this. */
__device__
void blattToStapp(hipDoubleComplex* phases,
				  hipDoubleComplex deltaMinusBB,
				  hipDoubleComplex deltaPlusBB,
				  hipDoubleComplex twoEpsilonJBB,
				  int TLabLength) {

	int slice = blockIdx.z * blockDim.z + threadIdx.z;
	if (slice < TLabLength) {
		hipDoubleComplex twoEpsilonJ = asinCudaComplex(sinCudaComplex(twoEpsilonJBB) * sinCudaComplex(deltaMinusBB - deltaPlusBB));

		phases[0 + slice*3] = 0.5 * (deltaPlusBB + deltaMinusBB + asinCudaComplex(tanCudaComplex(twoEpsilonJ) / tanCudaComplex(twoEpsilonJBB))) * constants::rad2deg;
		phases[1 + slice*3] = 0.5 * (deltaPlusBB + deltaMinusBB - asinCudaComplex(tanCudaComplex(twoEpsilonJ) / tanCudaComplex(twoEpsilonJBB))) * constants::rad2deg;
		phases[2 + slice*3] = 0.5 * twoEpsilonJ * constants::rad2deg;
	}
}


/**
	Computes the phase shift for a given channel and T matrix.

	@param channel: Scattering channel
	@param key:		Channel name
	@param k0:		On-shell-point
	@param T:		T matrix
	@return			Complex phase shifts
*/
__global__
void computePhaseShifts(hipDoubleComplex* phases,
	hipDoubleComplex* T,
	double* k0,
	int quadratureN,
	double mu,
	bool coupled,
	int TLabLength,
	int matLength) {

	int slice = blockIdx.x * blockDim.x + threadIdx.x;
	if (slice < TLabLength) {
		double rhoT = 2 * mu * k0[slice];
		const hipDoubleComplex I = make_hipDoubleComplex(0.0, 1.0);
		// TODO: Explain theory for the phase shift for the coupled state
		if (coupled) {
			hipDoubleComplex T11 = T[(quadratureN)+(quadratureN * quadratureN) + slice * matLength * matLength]; //row + column * size
			hipDoubleComplex T12 = T[(2 * quadratureN + 1) + (quadratureN * quadratureN) + slice * matLength * matLength];
			hipDoubleComplex T22 = T[(2 * quadratureN + 1) + (quadratureN * (2 * quadratureN + 1)) + slice * matLength * matLength];

			//Blatt - Biedenharn(BB) convention
			hipDoubleComplex twoEpsilonJBB = atanCudaComplex(hipCdiv(hipCmul(make_hipDoubleComplex(2.0, 0), T12), hipCsub(T11, T22)));
			hipDoubleComplex deltaPlusBB = -0.5 * I * logCudaComplex(1.0 - I * rhoT * (T11 + T22) + I * rhoT * (2.0 * T12) / sinCudaComplex(twoEpsilonJBB));
			hipDoubleComplex deltaMinusBB = -0.5 * I * logCudaComplex(1.0 - I * rhoT * (T11 + T22) - I * rhoT * (2.0 * T12) / sinCudaComplex(twoEpsilonJBB));

			blattToStapp(phases, deltaMinusBB, deltaPlusBB, twoEpsilonJBB, TLabLength);
		}
		/* The uncoupled case completely follows equation (2.26). */
		else {
			hipDoubleComplex T0 = (T[(quadratureN)+(quadratureN * matLength) + slice * matLength * matLength ]);
			hipDoubleComplex argument = make_hipDoubleComplex(1,0) - 2.0 * I * rhoT * T0;
			phases[slice] = -0.5 * I * constants::rad2deg * logCudaComplex(argument);
		}
	}
}