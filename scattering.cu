#include "hip/hip_runtime.h"
#include "scattering.h"

__device__
void cudaSetElement(hipDoubleComplex* tensor, int row, int column, int slice, int matLength, hipDoubleComplex value){
    tensor[row + column*matLength+slice*matLength*matLength] = value;
}

__device__
hipDoubleComplex cudaGetElement(hipDoubleComplex* tensor, int row, int column, int slice, int matLength){
    return tensor[row + column*matLength+slice*matLength*matLength];
}

__device__
hipDoubleComplex operator+(hipDoubleComplex A, hipDoubleComplex B) {
	hipDoubleComplex result = make_hipDoubleComplex(hipCreal(A)+hipCreal(B), hipCimag(A)+hipCimag(B));
	return result;
}

__device__
hipDoubleComplex operator-(hipDoubleComplex A, hipDoubleComplex B) {
	hipDoubleComplex result = make_hipDoubleComplex(hipCreal(A) - hipCreal(B), hipCimag(A) - hipCimag(B));
	return result;
}

__device__
hipDoubleComplex operator-(double a, hipDoubleComplex A) {
	hipDoubleComplex result = hipCsub(make_hipDoubleComplex(a, 0), A);
	return result;
}

__device__
hipDoubleComplex operator-(hipDoubleComplex A, double a) {
	hipDoubleComplex result = hipCsub(A, make_hipDoubleComplex(a, 0));
	return result;
}

__device__
hipDoubleComplex operator*(double scalar, hipDoubleComplex A) {
	hipDoubleComplex result = make_hipDoubleComplex(scalar * hipCreal(A), scalar * hipCimag(A));
	return result;
}

__device__
hipDoubleComplex operator*(hipDoubleComplex A, double scalar) {
	return scalar * A;
}

__device__
hipDoubleComplex operator*(hipDoubleComplex A, hipDoubleComplex B) {
	hipDoubleComplex realProd = hipCreal(A) * B;
	hipDoubleComplex imagProd = make_hipDoubleComplex(-hipCimag(A) * hipCimag(B), hipCimag(A)*hipCreal(B));
	hipDoubleComplex result = hipCadd(realProd, imagProd);
	return result;
}

__device__
hipDoubleComplex operator/(hipDoubleComplex A, hipDoubleComplex B) {
	return hipCdiv(A, B);
}

__device__
hipDoubleComplex operator/(hipDoubleComplex A, double a) {
	return hipCdiv(A, make_hipDoubleComplex(a, 0));
}

__device__
hipDoubleComplex operator/(double a, hipDoubleComplex A) {
	return hipCdiv(make_hipDoubleComplex(a, 0), A);
}

__device__
hipDoubleComplex logCudaComplex(hipDoubleComplex argument) {
	double x = hipCreal(argument);
	double y = hipCimag(argument);
	double real = logf(sqrtf(x * x + y * y));
	double imag = atan2f(y, x);
	hipDoubleComplex result = make_hipDoubleComplex(real, imag);
	return result;
}

__device__
double signCuda(double argument) {
	if (argument > 0) return 1;
	else if (argument == 0) return 0;
	else return -1;
}

__device__
hipDoubleComplex sqrtCudaComplex(hipDoubleComplex argument) {
	double x = hipCreal(argument);
	double y = hipCimag(argument);
	double real = sqrtf((sqrtf(x * x + y * y) + x) / 2);
	double imag = signCuda(y) * sqrtf((sqrtf(x * x + y * y) - x) / 2);
	return make_hipDoubleComplex(real, imag);
}

__device__
hipDoubleComplex atanCudaComplex(hipDoubleComplex argument) {
	hipDoubleComplex numerator = hipCadd(make_hipDoubleComplex(1, 0), hipCmul(make_hipDoubleComplex(0, 1), argument));
	hipDoubleComplex denominator = hipCsub(make_hipDoubleComplex(1, 0), hipCmul(make_hipDoubleComplex(0, 1), argument));
	hipDoubleComplex logOfStuff = logCudaComplex(hipCdiv(numerator, denominator));
	hipDoubleComplex result = hipCmul(make_hipDoubleComplex(0, -0.5), logOfStuff);
	return result;
}

__device__
hipDoubleComplex asinCudaComplex(hipDoubleComplex argument) {
	const hipDoubleComplex I = make_hipDoubleComplex(0.0, 1.0);
	return I * logCudaComplex(sqrtCudaComplex(1 - argument * argument) - I * argument);
}

__device__
hipDoubleComplex expCudaComplex(hipDoubleComplex argument) {
	double x = hipCreal(argument);
	double y = hipCimag(argument);
	hipDoubleComplex trig = make_hipDoubleComplex(cosf(y), sinf(y));
	hipDoubleComplex result = make_hipDoubleComplex(expf(x), 0) * trig;
	return result;
}

__device__
hipDoubleComplex sinCudaComplex(hipDoubleComplex argument) {
	const hipDoubleComplex I = make_hipDoubleComplex(0.0, 1.0);
	return (expCudaComplex(I * argument) - expCudaComplex(-1.0 * I * argument)) / 2;
}

__device__
hipDoubleComplex tanCudaComplex(hipDoubleComplex argument) {
	const hipDoubleComplex I = make_hipDoubleComplex(0.0, 1.0);
	hipDoubleComplex numerator = I * (expCudaComplex(-1.0 * I * argument) - expCudaComplex(I * argument));
	hipDoubleComplex denominator = expCudaComplex(-1.0 * I * argument) + expCudaComplex(I * argument);
	return numerator / denominator;
}

__global__
void setupG0Vector(hipDoubleComplex* G0,
	double* k,
	double* w,
	double* k0,
	double* sum,
	int quadratureN,
	int matLength,
	int TLabLength,
	double mu,
	bool coupled) {

	int column = blockIdx.x * blockDim.x + threadIdx.x;
	int slice = blockIdx.z * blockDim.z + threadIdx.z;

	double twoMu = (2.0 * mu);
	double twoOverPi = (2.0 / constants::pi);

	if (column < quadratureN && slice < TLabLength) {
		G0[column + slice * matLength] = make_hipDoubleComplex(twoOverPi * twoMu * k[column] * k[column] * w[column] / (k0[slice] * k0[slice] - k[column] * k[column]), 0);

		/* If coupled, append G0 to itself to facilitate calculations.
		 * This means the second half of G0 is a copy of the first. */
		if (coupled) {
			G0[quadratureN + 1 + column + slice * matLength] = G0[column + slice * matLength];
		}

		/* Assign the last element of D */
		G0[quadratureN + slice * matLength] = make_hipDoubleComplex(-twoOverPi * twoMu * k0[slice] * k0[slice] * sum[slice], -twoMu * k0[slice]);
		if (coupled) {
			G0[2 * (quadratureN + 1) - 1 + slice * matLength] = G0[quadratureN + slice * matLength];
		}

		printf("\nG0[col = %i, sli = %i] = %.4e, imag = %.4e\n", column+1, slice, hipCreal(G0[column+1 + slice * matLength]), hipCimag(G0[column+1 + slice * matLength]));
	}
}

/**
	Multiplies the potential matrix elements with the G0 vector elements.

	@param channel: Scattering channel
	@param key:		Channel name
	@param V:		Potential matrix
	@param k:		Quadrature points
	@param w:		Weights for quadrature points
	@param k0:		The on-shell-point
	@return			VG kernel
*/
__global__
void setupVGKernel(hipDoubleComplex* VG,
	hipDoubleComplex* V,
	hipDoubleComplex* G0,
	hipDoubleComplex* F,
	double* k,
	double* w,
	double* k0,
	int quadratureN,
	int matLength,
	int TLabLength,
	double mu,
	bool coupled) {

	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int column = blockIdx.x * blockDim.x + threadIdx.x;
	int slice = blockIdx.z * blockDim.z + threadIdx.z;

	if (row < matLength && column < matLength && slice < TLabLength) {
		VG[row + column * matLength + slice * matLength * matLength] = hipCmul(V[row + column * matLength + slice * matLength * matLength], G0[column + slice * matLength]);

		if (row == column) {
			F[row + row * matLength + slice * matLength * matLength] = hipCadd(make_hipDoubleComplex(1, 0), hipCmul(make_hipDoubleComplex(-1, 0), VG[row + row * matLength + slice * matLength * matLength])); // Diagonal element
		}
		else {
			F[row + column * matLength + slice * matLength * matLength] = hipCmul(make_hipDoubleComplex(-1, 0), VG[row + column * matLength + slice * matLength * matLength]);
		}
		printf("\nF[col = %i, row = %i, sli = %i] = %.4e\n", column, row, slice, hipCreal(F[row + column * matLength+slice * matLength * matLength]));
	

	}

}



	//for (int row = 0; row < matLength; row++) {
	//	for (int column = 0; column < matLength; column++) {
	//		/* Create VG by using VG[i,j] = V[i,j] * G[j] */
	//		VG[row + column * matLength] = hipCmul(V[row + column * matLength], G0[column]);

	//		/* At the same time, create F = delta_ij - VG_ij for computeTMatrix*/
	//		if (row != column) {
	//			F[row + column * matLength] = hipCmul(make_hipDoubleComplex(-1, 0), VG[row + column * matLength]);
	//		}
	//	}
	//	F[row + row * matLength] = hipCadd(make_hipDoubleComplex(1, 0), hipCmul(make_hipDoubleComplex(-1, 0), VG[row + row * matLength])); // Diagonal element
	//}
//}




/**
	Computes the T-matrix from the equation [F][T] = [V]

	@param channel: Scattering channel
	@param key:		Channel name
	@param V:		Potential matrix
	@param k:		Quadrature points
	@param w:		Weights for quadrature points
	@param k0:		On-shell-point
	@return			T matrix
*/


/* TODO: Explain theory for this. */
__device__
void blattToStapp(hipDoubleComplex* phases,
				  hipDoubleComplex* deltaMinusBB,
				  hipDoubleComplex* deltaPlusBB,
				  hipDoubleComplex* twoEpsilonJBB,
				  int TLabLength) {

	int slice = blockIdx.z * blockDim.z + threadIdx.z;
	hipDoubleComplex* twoEpsilonJ = new hipDoubleComplex[TLabLength];
	if (slice < TLabLength) {
		twoEpsilonJ[slice] = asinCudaComplex(sinCudaComplex(twoEpsilonJBB[slice]) * sinCudaComplex(deltaMinusBB[slice] - deltaPlusBB[slice]));

		phases[0 + slice*3] = 0.5 * (deltaPlusBB[slice] + deltaMinusBB[slice] + asinCudaComplex(tanCudaComplex(twoEpsilonJ[slice]) / tanCudaComplex(twoEpsilonJBB[slice]))) * constants::rad2deg;
		phases[1 + slice*3] = 0.5 * (deltaPlusBB[slice] + deltaMinusBB[slice] - asinCudaComplex(tanCudaComplex(twoEpsilonJ[slice]) / tanCudaComplex(twoEpsilonJBB[slice]))) * constants::rad2deg;
		phases[2 + slice*3] = 0.5 * twoEpsilonJ[slice] * constants::rad2deg;
	}
}


/**
	Computes the phase shift for a given channel and T matrix.

	@param channel: Scattering channel
	@param key:		Channel name
	@param k0:		On-shell-point
	@param T:		T matrix
	@return			Complex phase shifts
*/
__global__
void computePhaseShifts(hipDoubleComplex* phases,
	hipDoubleComplex* T,
	double* k0,
	int quadratureN,
	double mu,
	bool coupled,
	int TLabLength,
	int matLength) {

	int slice = blockIdx.x * blockDim.x + threadIdx.x;

	double* rhoT = new double[TLabLength];
	hipDoubleComplex* T11 = new hipDoubleComplex[TLabLength];
	hipDoubleComplex* T12 = new hipDoubleComplex[TLabLength];
	hipDoubleComplex* T22 = new hipDoubleComplex[TLabLength];
	hipDoubleComplex* twoEpsilonJBB = new hipDoubleComplex[TLabLength];
	hipDoubleComplex* deltaPlusBB = new hipDoubleComplex[TLabLength];
	hipDoubleComplex* deltaMinusBB = new hipDoubleComplex[TLabLength];

	if (slice < TLabLength) {

		rhoT[slice] = 2 * mu * k0[slice]; // Equation (2.27) in the theory
		const hipDoubleComplex I = make_hipDoubleComplex(0.0, 1.0);

		// TODO: Explain theory for the phase shift for the coupled state
		if (coupled) {
			T11[slice] = T[(quadratureN)+(quadratureN * quadratureN) + slice * matLength * matLength]; //row + column * size
			T12[slice] = T[(2 * quadratureN + 1) + (quadratureN * quadratureN) + slice * matLength * matLength];
			T22[slice] = T[(2 * quadratureN + 1) + (quadratureN * (2 * quadratureN + 1)) + slice * matLength * matLength];

			//Blatt - Biedenharn(BB) convention
			twoEpsilonJBB[slice] = atanCudaComplex(hipCdiv(hipCmul(make_hipDoubleComplex(2.0, 0), T12[slice]), hipCsub(T11[slice], T22[slice])));
			deltaPlusBB[slice] = -0.5 * I * logCudaComplex(1.0 - I * rhoT[slice] * (T11[slice] + T22[slice]) + I * rhoT[slice] * (2.0 * T12[slice]) / sinCudaComplex(twoEpsilonJBB[slice]));
			deltaMinusBB[slice] = -0.5 * I * logCudaComplex(1.0 - I * rhoT[slice] * (T11[slice] + T22[slice]) - I * rhoT[slice] * (2.0 * T12[slice]) / sinCudaComplex(twoEpsilonJBB[slice]));

		}
		/* The uncoupled case completely follows equation (2.26). */
		else {
			hipDoubleComplex T0 = (T[(quadratureN)+(quadratureN * matLength) + slice * matLength * matLength ]); //Farligt, detta element kanske inte �r helt reellt. Dock var koden d�lig f�rut is�fall.
			printf("\ngrej = %.4e, imag = %.4e\n", hipCreal(2.0 * rhoT[slice] * T0 * I), hipCimag(2.0 * rhoT[slice] * T0 * I));
			hipDoubleComplex* argument = new hipDoubleComplex[TLabLength];
			argument[slice] = make_hipDoubleComplex(1,0) - 2.0 * I * rhoT[slice] * T0;
			printf("\nargument[slice = %i] = %.4e, imag = %.4e\n", slice, hipCreal(argument[slice]), hipCimag(argument[slice]));
			hipDoubleComplex swappedLog = make_hipDoubleComplex(hipCimag(logCudaComplex(argument[slice])), hipCreal(logCudaComplex(argument[slice])));
			hipDoubleComplex delta = hipCmul(make_hipDoubleComplex(-0.5 * constants::rad2deg, 0), swappedLog);
			phases[slice] = delta;
		}
	}

	if (coupled) {
		blattToStapp(phases, deltaMinusBB, deltaPlusBB, twoEpsilonJBB, TLabLength);
	}
}