#include "hip/hip_runtime.h"
#include "scattering.h"


/**
	Sets up a complex vector needed to solve the T matrix equation.
	@param k:	Quadrature points
	@param w:	Weights for quadrature points
	@param k0:	On-shell-point
	@return		G0 vector
*/

//const hipDoubleComplex I = make_hipDoubleComplex(0.0, 1.0);


template <typename T>
void check(T result, char const* const func, const char* const file,
	int const line) {
	if (result) {
		fprintf(stderr, "CUDA error at %s:%d code=%d(%s) \"%s\" \n", file, line,
			static_cast<unsigned int>(result), _cudaGetErrorEnum(result), func);
		exit(EXIT_FAILURE);
	}
}

#define checkCudaErrors(val) check((val), #val, __FILE__, __LINE__)

static const char* _cudaGetErrorEnum(hipError_t error) {
	return hipGetErrorName(error);
}

__device__
void computeTMatrixCUBLAS(hipDoubleComplex* h_Tarray,
	hipDoubleComplex* h_Farray,
	hipDoubleComplex* h_Varray,
	int N, int mSize) {

	const int batchSize{1};
	if (true) {
		const int batchSize = mSize;
	}

	// cuBLAS variables
	hipblasStatus_t status;
	hipblasHandle_t handle;

	// Host variables
	size_t matSize = N * N * sizeof(hipDoubleComplex); //denna ska la dock vara mSize * mSize, ty matriserna som löses har den storleken.

	// hipDoubleComplex* h_Farray;
	hipDoubleComplex* h_Fptr_array[batchSize];

	// hipDoubleComplex* h_Varray;
	hipDoubleComplex* h_Vptr_array[batchSize];

	// Device variables
	hipDoubleComplex* d_Farray;
	hipDoubleComplex** d_Fptr_array;

	hipDoubleComplex* d_Varray;
	hipDoubleComplex** d_Vptr_array;

	int* d_pivotArray;
	int* d_trfInfo;
	int d_trsInfo;

	// Initialize cuBLAS
	status = hipblasCreate(&handle);
	if (status != HIPBLAS_STATUS_SUCCESS) {
		printf("> ERROR: cuBLAS initialization failed\n");
		//return (EXIT_FAILURE); /return type är void, inte exit_failure
	}

	// Allocate memory for host variables
	// h_Farray = (hipDoubleComplex*)xmalloc(batchSize * matSize);
	// h_Varray = (hipDoubleComplex*)xmalloc(batchSize * matSize);

	// Allocate memory for device variables

	checkCudaErrors(hipMalloc((void**)&d_Farray, batchSize * matSize));
	checkCudaErrors(hipMalloc((void**)&d_Varray, batchSize * matSize));
	checkCudaErrors(
		hipMalloc((void**)&d_pivotArray, N * batchSize * sizeof(int)));
	checkCudaErrors(hipMalloc((void**)&d_trfInfo, batchSize * sizeof(int)));
	checkCudaErrors(
		hipMalloc((void**)&d_Fptr_array, batchSize * sizeof(hipDoubleComplex*)));
	checkCudaErrors(
		hipMalloc((void**)&d_Vptr_array, batchSize * sizeof(hipDoubleComplex*)));

	// for (int i = 0; i < batchSize; i++) {
	//     initSetAMatrix(h_Farray + (i * N*N), (double)(i+1)); // Create matrices scaled by factors 1, 2, ...
	// }

	// printMatrix(h_Farray, N, N);

	// for (int i = 0; i < batchSize; i++) {
	//     initSetBMatrix(h_Varray + (i * N*N), (double)(i+1)); // Create matrices scaled by factors 1, 2, ...
	// }

	// printMatrix(h_Varray, N, N);

	// Copy data to device from host
	checkCudaErrors(hipMemcpy(d_Farray, h_Farray, batchSize * matSize,
		hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_Varray, h_Varray, batchSize * matSize,
		hipMemcpyHostToDevice));

	// Create pointer array for matrices
	for (int i = 0; i < batchSize; i++) h_Fptr_array[i] = d_Farray + (i * N * N);
	for (int i = 0; i < batchSize; i++) h_Vptr_array[i] = d_Varray + (i * N * N);

	// Copy pointer array to device memory
	checkCudaErrors(hipMemcpy(d_Fptr_array, h_Fptr_array,
		batchSize * sizeof(hipDoubleComplex*),
		hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_Vptr_array, h_Vptr_array,
		batchSize * sizeof(hipDoubleComplex*),
		hipMemcpyHostToDevice));

	// Perform LU decomposition
	status = hipblasZgetrfBatched(handle, N, d_Fptr_array, N, d_pivotArray,
		d_trfInfo, batchSize);

	// Calculate the T matrix
	status = hipblasZgetrsBatched(handle, HIPBLAS_OP_N, N, N, d_Fptr_array, N,
		d_pivotArray, d_Vptr_array, N, &d_trsInfo,
		batchSize);

	// Copy data to host from device
	checkCudaErrors(hipMemcpy(h_Tarray, d_Varray, batchSize * matSize,
		hipMemcpyDeviceToHost));

	// printMatrix(h_Varray, N, N);

	// Free device variables
	checkCudaErrors(hipFree(d_Fptr_array));
	checkCudaErrors(hipFree(d_Vptr_array));
	checkCudaErrors(hipFree(d_trfInfo));
	checkCudaErrors(hipFree(d_pivotArray));
	checkCudaErrors(hipFree(d_Farray));
	checkCudaErrors(hipFree(d_Varray));

	// Free host variables
	// if (h_Farray) free(h_Farray);
	// if (h_Varray) free(h_Varray);

	// Destroy cuBLAS handle
	status = hipblasDestroy(handle);
	if (status != HIPBLAS_STATUS_SUCCESS) {
		printf("> ERROR: cuBLAS uninitialization failed...\n");
	}
}








__device__
hipDoubleComplex operator+(hipDoubleComplex A, hipDoubleComplex B) {
	hipDoubleComplex result = make_hipDoubleComplex(hipCreal(A)+hipCreal(B), hipCimag(A)+hipCimag(B));
	return result;
}

__device__
hipDoubleComplex operator-(hipDoubleComplex A, hipDoubleComplex B) {
	hipDoubleComplex result = make_hipDoubleComplex(hipCreal(A) - hipCreal(B), hipCimag(A) - hipCimag(B));
	return result;
}

__device__
hipDoubleComplex operator-(double a, hipDoubleComplex A) {
	hipDoubleComplex result = hipCsub(make_hipDoubleComplex(a, 0), A);
	return result;
}

__device__
hipDoubleComplex operator-(hipDoubleComplex A, double a) {
	hipDoubleComplex result = hipCsub(A, make_hipDoubleComplex(a, 0));
	return result;
}

__device__
hipDoubleComplex operator*(double scalar, hipDoubleComplex A) {
	hipDoubleComplex result = make_hipDoubleComplex(scalar * hipCreal(A), scalar * hipCimag(A));
	return result;
}

__device__
hipDoubleComplex operator*(hipDoubleComplex A, double scalar) {
	return scalar * A;
}

__device__
hipDoubleComplex operator*(hipDoubleComplex A, hipDoubleComplex B) {
	hipDoubleComplex realProd = hipCreal(A) * B;
	hipDoubleComplex imagProd = hipCimag(A) * B;
	hipDoubleComplex result = hipCadd(realProd, imagProd);
	return result;
}

__device__
hipDoubleComplex operator/(hipDoubleComplex A, hipDoubleComplex B) {
	return hipCdiv(A, B);
}

__device__
hipDoubleComplex operator/(hipDoubleComplex A, double a) {
	return hipCdiv(A, make_hipDoubleComplex(a, 0));
}

__device__
hipDoubleComplex operator/(double a, hipDoubleComplex A) {
	return hipCdiv(make_hipDoubleComplex(a, 0), A);
}

__device__
hipDoubleComplex logCudaComplex(hipDoubleComplex argument) {
	double x = hipCreal(argument);
	double y = hipCimag(argument);
	double real = logf(sqrtf(x * x + y * y));
	double imag = atan2f(y, x);
	hipDoubleComplex result = make_hipDoubleComplex(real, imag);
	return result;
}

__device__
double signCuda(double argument) {
	if (argument > 0) return 1;
	else if (argument == 0) return 0;
	else return -1;
}

__device__
hipDoubleComplex sqrtCudaComplex(hipDoubleComplex argument) {
	double x = hipCreal(argument);
	double y = hipCimag(argument);
	double real = sqrtf((sqrtf(x * x + y * y) + x) / 2);
	double imag = signCuda(y) * sqrtf((sqrtf(x * x + y * y) - x) / 2);
	return make_hipDoubleComplex(real, imag);
}

__device__
hipDoubleComplex atanCudaComplex(hipDoubleComplex argument) {
	hipDoubleComplex numerator = hipCadd(make_hipDoubleComplex(1, 0), hipCmul(make_hipDoubleComplex(0, 1), argument));
	hipDoubleComplex denominator = hipCsub(make_hipDoubleComplex(1, 0), hipCmul(make_hipDoubleComplex(0, 1), argument));
	hipDoubleComplex logOfStuff = logCudaComplex(hipCdiv(numerator, denominator));
	hipDoubleComplex result = hipCmul(make_hipDoubleComplex(0, -0.5), logOfStuff);
	return result;
}

__device__
hipDoubleComplex asinCudaComplex(hipDoubleComplex argument) {
	const hipDoubleComplex I = make_hipDoubleComplex(0.0, 1.0);
	return I * logCudaComplex(sqrtCudaComplex(1 - argument * argument) - I * argument);
}

__device__
hipDoubleComplex expCudaComplex(hipDoubleComplex argument) {
	double x = hipCreal(argument);
	double y = hipCimag(argument);
	hipDoubleComplex trig = make_hipDoubleComplex(cosf(y), sinf(y));
	hipDoubleComplex result = make_hipDoubleComplex(expf(x), 0) * trig;
	return result;
}

__device__
hipDoubleComplex sinCudaComplex(hipDoubleComplex argument) {
	const hipDoubleComplex I = make_hipDoubleComplex(0.0, 1.0);
	return (expCudaComplex(I * argument) - expCudaComplex(-1.0 * I * argument)) / 2;
}

__device__
hipDoubleComplex tanCudaComplex(hipDoubleComplex argument) {
	const hipDoubleComplex I = make_hipDoubleComplex(0.0, 1.0);
	hipDoubleComplex numerator = I * (expCudaComplex(-1.0 * I * argument) - expCudaComplex(I * argument));
	hipDoubleComplex denominator = expCudaComplex(-1.0 * I * argument) + expCudaComplex(I * argument);
	return numerator / denominator;
}




__device__
void setupG0Vector(hipDoubleComplex* G0,
	double* k,
	double* w,
	double k0,
	int quadratureN,
	double mu,
	bool coupled) {

	double twoMu = (2.0 * mu);
	double twoOverPi = (2.0 / constants::pi);
	double sum = 0;
	for (int i = 0; i < quadratureN; i++) {
		G0[i] = make_hipDoubleComplex(twoOverPi * twoMu * k[i] * k[i] * w[i] / (k0 * k0 - k[i] * k[i]), 0);
		sum += w[i] / (k0 * k0 - k[i] * k[i]);

		/* If coupled, append G0 to itself to facilitate calculations.
		 * This means the second half of G0 is a copy of the first. */
		if (coupled) {
			G0[quadratureN + 1 + i] = G0[i];
		}
	}

	/* Assign the last element of D */
	G0[quadratureN] = make_hipDoubleComplex(-twoOverPi * twoMu * k0 * k0 * sum, -twoMu * k0);
	if (coupled) {
		G0[2 * (quadratureN + 1) - 1] = G0[quadratureN];
	}
}

/**
	Multiplies the potential matrix elements with the G0 vector elements.

	@param channel: Scattering channel
	@param key:		Channel name
	@param V:		Potential matrix
	@param k:		Quadrature points
	@param w:		Weights for quadrature points
	@param k0:		The on-shell-point
	@return			VG kernel
*/
__device__
void setupVGKernel(hipDoubleComplex* VG,
	hipDoubleComplex* V,
	hipDoubleComplex* G0,
	hipDoubleComplex* F,
	double* k,
	double* w,
	double k0,
	int quadratureN,
	int matSize,
	double mu,
	bool coupled) {

	setupG0Vector(G0, k, w, k0, quadratureN, mu, coupled);
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int column = blockIdx.x * blockDim.x + threadIdx.x;

	if (row < matSize && column < matSize) {
		VG[row + column * matSize] = hipCmul(V[row + column * matSize], G0[column]);
		//hipDoubleComplex test = hipCmul(V[row + column * matSize], G0[column]);
		hipDoubleComplex testG0 = G0[column];
		hipDoubleComplex testV = V[row + column * matSize];
		//printf("\nV = %f, %f", hipCreal(testV), hipCimag(testV));
		if (row == column) {
			F[row + row * matSize] = hipCadd(make_hipDoubleComplex(1, 0), hipCmul(make_hipDoubleComplex(-1, 0), VG[row + row * matSize])); // Diagonal element
		}
		else {
			F[row + column * matSize] = hipCmul(make_hipDoubleComplex(-1, 0), VG[row + column * matSize]);
		}

	}

}



	//for (int row = 0; row < matSize; row++) {
	//	for (int column = 0; column < matSize; column++) {
	//		/* Create VG by using VG[i,j] = V[i,j] * G[j] */
	//		VG[row + column * matSize] = hipCmul(V[row + column * matSize], G0[column]);

	//		/* At the same time, create F = delta_ij - VG_ij for computeTMatrix*/
	//		if (row != column) {
	//			F[row + column * matSize] = hipCmul(make_hipDoubleComplex(-1, 0), VG[row + column * matSize]);
	//		}
	//	}
	//	F[row + row * matSize] = hipCadd(make_hipDoubleComplex(1, 0), hipCmul(make_hipDoubleComplex(-1, 0), VG[row + row * matSize])); // Diagonal element
	//}
//}




/**
	Computes the T-matrix from the equation [F][T] = [V]

	@param channel: Scattering channel
	@param key:		Channel name
	@param V:		Potential matrix
	@param k:		Quadrature points
	@param w:		Weights for quadrature points
	@param k0:		On-shell-point
	@return			T matrix
*/
__global__
void computeTMatrix(hipDoubleComplex** T,
	hipDoubleComplex** V,
	hipDoubleComplex** G0,
	hipDoubleComplex** VG,
	hipDoubleComplex** F,
	hipDoubleComplex** phases,
	double* k,
	double* w,
	double* k0,
	int quadratureN,
	int matSize,
	int TLabLength,
	double mu,
	bool coupled) {

	/* Setup the VG kernel and, at the same time, the F matrix */
	for (int i = 0; i < TLabLength; i++) {
		setupVGKernel(VG[i], V[i], G0[i], F[i], k, w, k0[i], quadratureN, matSize, mu, coupled);

		/* Solve the equation FT = V with cuBLAS */
		computeTMatrixCUBLAS(T[i], F[i], V[i], quadratureN, matSize);

		/* Computes the phase shifts for the given T-matrix*/
		computePhaseShifts(phases[i], T[i], k0[i], quadratureN, mu, coupled);
	}

}



/* TODO: Explain theory for this. */
__device__
void blattToStapp(hipDoubleComplex* phases, hipDoubleComplex deltaMinusBB, hipDoubleComplex deltaPlusBB, hipDoubleComplex twoEpsilonJBB) {
	hipDoubleComplex twoEpsilonJ = asinCudaComplex(sinCudaComplex(twoEpsilonJBB) * sinCudaComplex(deltaMinusBB - deltaPlusBB));

	phases[0] = 0.5 * (deltaPlusBB + deltaMinusBB + asinCudaComplex(tanCudaComplex(twoEpsilonJ) / tanCudaComplex(twoEpsilonJBB))) * constants::rad2deg;
	phases[1] = 0.5 * (deltaPlusBB + deltaMinusBB - asinCudaComplex(tanCudaComplex(twoEpsilonJ) / tanCudaComplex(twoEpsilonJBB))) * constants::rad2deg;
	phases[2] = 0.5 * twoEpsilonJ * constants::rad2deg;
}


/**
	Computes the phase shift for a given channel and T matrix.

	@param channel: Scattering channel
	@param key:		Channel name
	@param k0:		On-shell-point
	@param T:		T matrix
	@return			Complex phase shifts
*/
__device__
void computePhaseShifts(hipDoubleComplex* phases,
					    hipDoubleComplex* T,
						double k0,
						int quadratureN,
						double mu,
						bool coupled) {

	double rhoT =  2 * mu * k0; // Equation (2.27) in the theory
	const hipDoubleComplex I = make_hipDoubleComplex(0.0, 1.0);

	// TODO: Explain theory for the phase shift for the coupled state
	if (coupled) {
		int N = quadratureN;
		hipDoubleComplex T11 = T[(N) + (N * N)]; //row + column * size
		hipDoubleComplex T12 = T[(2 * N + 1) + (N * N)];
		hipDoubleComplex T22 = T[(2 * N + 1) + (N * (2 * N + 1))];

		//Blatt - Biedenharn(BB) convention
		hipDoubleComplex twoEpsilonJBB = atanCudaComplex(hipCdiv(hipCmul(make_hipDoubleComplex(2.0, 0), T12), hipCsub(T11, T22)));
		hipDoubleComplex deltaPlusBB{ -0.5 * I * logCudaComplex(1.0 - I * rhoT * (T11 + T22) + I * rhoT * (2.0 * T12) / sinCudaComplex(twoEpsilonJBB)) };
		hipDoubleComplex deltaMinusBB{ -0.5 * I * logCudaComplex(1.0 - I * rhoT * (T11 + T22) - I * rhoT * (2.0 * T12) / sinCudaComplex(twoEpsilonJBB)) };
		blattToStapp(phases, deltaMinusBB, deltaPlusBB, twoEpsilonJBB);

	}
	/* The uncoupled case completely follows equation (2.26). */
	else {
		double T0 = hipCreal(T[(quadratureN) + (quadratureN * quadratureN)]); //Farligt, detta element kanske inte �r helt reellt. Dock var koden d�lig f�rut is�fall.
		hipDoubleComplex argument = make_hipDoubleComplex(1, -2.0 * rhoT * T0);
		hipDoubleComplex swappedLog = make_hipDoubleComplex(hipCimag(logCudaComplex(argument)), hipCreal(logCudaComplex(argument)));
		hipDoubleComplex delta = hipCmul(make_hipDoubleComplex(-0.5 * constants::rad2deg, 0), swappedLog);
		phases[0] = delta;
	}
}