#include "hip/hip_runtime.h"
#include "scattering.h"


/**
	Sets up a complex vector needed to solve the T matrix equation.
	@param k:	Quadrature points
	@param w:	Weights for quadrature points
	@param k0:	On-shell-point
	@return		G0 vector
*/

//const hipDoubleComplex I = make_hipDoubleComplex(0.0, 1.0);


__device__
hipDoubleComplex operator+(hipDoubleComplex A, hipDoubleComplex B) {
	hipDoubleComplex result = make_hipDoubleComplex(hipCreal(A)+hipCreal(B), hipCimag(A)+hipCimag(B));
	return result;
}

__device__
hipDoubleComplex operator-(hipDoubleComplex A, hipDoubleComplex B) {
	hipDoubleComplex result = make_hipDoubleComplex(hipCreal(A) - hipCreal(B), hipCimag(A) - hipCimag(B));
	return result;
}

__device__
hipDoubleComplex operator-(double a, hipDoubleComplex A) {
	hipDoubleComplex result = hipCsub(make_hipDoubleComplex(a, 0), A);
	return result;
}

__device__
hipDoubleComplex operator-(hipDoubleComplex A, double a) {
	hipDoubleComplex result = hipCsub(A, make_hipDoubleComplex(a, 0));
	return result;
}

__device__
hipDoubleComplex operator*(double scalar, hipDoubleComplex A) {
	hipDoubleComplex result = make_hipDoubleComplex(scalar * hipCreal(A), scalar * hipCimag(A));
	return result;
}

__device__
hipDoubleComplex operator*(hipDoubleComplex A, double scalar) {
	return scalar * A;
}

__device__
hipDoubleComplex operator*(hipDoubleComplex A, hipDoubleComplex B) {
	hipDoubleComplex realProd = hipCreal(A) * B;
	hipDoubleComplex imagProd = hipCimag(A) * B;
	hipDoubleComplex result = hipCadd(realProd, imagProd);
	return result;
}

__device__
hipDoubleComplex operator/(hipDoubleComplex A, hipDoubleComplex B) {
	return hipCdiv(A, B);
}

__device__
hipDoubleComplex operator/(hipDoubleComplex A, double a) {
	return hipCdiv(A, make_hipDoubleComplex(a, 0));
}

__device__
hipDoubleComplex operator/(double a, hipDoubleComplex A) {
	return hipCdiv(make_hipDoubleComplex(a, 0), A);
}

__device__
hipDoubleComplex logCudaComplex(hipDoubleComplex argument) {
	double x = hipCreal(argument);
	double y = hipCimag(argument);
	double real = logf(sqrtf(x * x + y * y));
	double imag = atan2f(y, x);
	hipDoubleComplex result = make_hipDoubleComplex(real, imag);
	return result;
}

__device__
double signCuda(double argument) {
	if (argument > 0) return 1;
	else if (argument == 0) return 0;
	else return -1;
}

__device__
hipDoubleComplex sqrtCudaComplex(hipDoubleComplex argument) {
	double x = hipCreal(argument);
	double y = hipCimag(argument);
	double real = sqrtf((sqrtf(x * x + y * y) + x) / 2);
	double imag = signCuda(y) * sqrtf((sqrtf(x * x + y * y) - x) / 2);
	return make_hipDoubleComplex(real, imag);
}

__device__
hipDoubleComplex atanCudaComplex(hipDoubleComplex argument) {
	hipDoubleComplex numerator = hipCadd(make_hipDoubleComplex(1, 0), hipCmul(make_hipDoubleComplex(0, 1), argument));
	hipDoubleComplex denominator = hipCsub(make_hipDoubleComplex(1, 0), hipCmul(make_hipDoubleComplex(0, 1), argument));
	hipDoubleComplex logOfStuff = logCudaComplex(hipCdiv(numerator, denominator));
	hipDoubleComplex result = hipCmul(make_hipDoubleComplex(0, -0.5), logOfStuff);
	return result;
}

__device__
hipDoubleComplex asinCudaComplex(hipDoubleComplex argument) {
	const hipDoubleComplex I = make_hipDoubleComplex(0.0, 1.0);
	return I * logCudaComplex(sqrtCudaComplex(1 - argument * argument) - I * argument);
}

__device__
hipDoubleComplex expCudaComplex(hipDoubleComplex argument) {
	double x = hipCreal(argument);
	double y = hipCimag(argument);
	hipDoubleComplex trig = make_hipDoubleComplex(cosf(y), sinf(y));
	hipDoubleComplex result = make_hipDoubleComplex(expf(x), 0) * trig;
	return result;
}

__device__
hipDoubleComplex sinCudaComplex(hipDoubleComplex argument) {
	const hipDoubleComplex I = make_hipDoubleComplex(0.0, 1.0);
	return (expCudaComplex(I * argument) - expCudaComplex(-1.0 * I * argument)) / 2;
}

__device__
hipDoubleComplex tanCudaComplex(hipDoubleComplex argument) {
	const hipDoubleComplex I = make_hipDoubleComplex(0.0, 1.0);
	hipDoubleComplex numerator = I * (expCudaComplex(-1.0 * I * argument) - expCudaComplex(I * argument));
	hipDoubleComplex denominator = expCudaComplex(-1.0 * I * argument) + expCudaComplex(I * argument);
	return numerator / denominator;
}




__global__
void setupG0Vector(hipDoubleComplex* G0,
	double* k,
	double* w,
	double* k0,
	int quadratureN,
	int matLength,
	int TLabLength,
	double mu,
	bool coupled) {

	int column = blockIdx.x * blockDim.x + threadIdx.x;
	int width = blockIdx.z * blockDim.z + threadIdx.z;

	double twoMu = (2.0 * mu);
	double twoOverPi = (2.0 / constants::pi);
	double sum = 0;

	if (column < quadratureN && width < TLabLength) {
		G0[column + width * matLength] = make_hipDoubleComplex(twoOverPi * twoMu * k[column] * k[column] * w[column] / (k0[width] * k0[width] - k[column] * k[column]), 0);
		sum += w[column] / (k0[width] * k0[width] - k[column] * k[column]);

		/* If coupled, append G0 to itself to facilitate calculations.
		 * This means the second half of G0 is a copy of the first. */
		if (coupled) {
			G0[quadratureN + 1 + column + width * matLength] = G0[column + width * matLength];
		}

		/* Assign the last element of D */
		G0[quadratureN + width * matLength] = make_hipDoubleComplex(-twoOverPi * twoMu * k0[width] * k0[width] * sum, -twoMu * k0[width]);
		if (coupled) {
			G0[2 * (quadratureN + 1) - 1 + width * matLength] = G0[quadratureN + width * matLength];
		}
	}
}

/**
	Multiplies the potential matrix elements with the G0 vector elements.

	@param channel: Scattering channel
	@param key:		Channel name
	@param V:		Potential matrix
	@param k:		Quadrature points
	@param w:		Weights for quadrature points
	@param k0:		The on-shell-point
	@return			VG kernel
*/
__global__
void setupVGKernel(hipDoubleComplex* VG,
	hipDoubleComplex* V,
	hipDoubleComplex* G0,
	hipDoubleComplex* F,
	double* k,
	double* w,
	double* k0,
	int quadratureN,
	int matLength,
	int TLabLength,
	double mu,
	bool coupled) {

	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int column = blockIdx.x * blockDim.x + threadIdx.x;
	int width = blockIdx.z * blockDim.z + threadIdx.z;

	if (row < matLength && column < matLength && width < TLabLength) {
		VG[row + column * matLength + width * matLength * matLength] = hipCmul(V[row + column * matLength + width * matLength * matLength], G0[column + width * matLength]);

		if (row == column) {
			F[row + row * matLength + width * matLength * matLength] = hipCadd(make_hipDoubleComplex(1, 0), hipCmul(make_hipDoubleComplex(-1, 0), VG[row + row * matLength + width * matLength * matLength])); // Diagonal element
		}
		else {
			F[row + column * matLength + width * matLength * matLength] = hipCmul(make_hipDoubleComplex(-1, 0), VG[row + column * matLength + width * matLength * matLength]);
		}

	}

}



	//for (int row = 0; row < matLength; row++) {
	//	for (int column = 0; column < matLength; column++) {
	//		/* Create VG by using VG[i,j] = V[i,j] * G[j] */
	//		VG[row + column * matLength] = hipCmul(V[row + column * matLength], G0[column]);

	//		/* At the same time, create F = delta_ij - VG_ij for computeTMatrix*/
	//		if (row != column) {
	//			F[row + column * matLength] = hipCmul(make_hipDoubleComplex(-1, 0), VG[row + column * matLength]);
	//		}
	//	}
	//	F[row + row * matLength] = hipCadd(make_hipDoubleComplex(1, 0), hipCmul(make_hipDoubleComplex(-1, 0), VG[row + row * matLength])); // Diagonal element
	//}
//}




/**
	Computes the T-matrix from the equation [F][T] = [V]

	@param channel: Scattering channel
	@param key:		Channel name
	@param V:		Potential matrix
	@param k:		Quadrature points
	@param w:		Weights for quadrature points
	@param k0:		On-shell-point
	@return			T matrix
*/


/* TODO: Explain theory for this. */
__device__
void blattToStapp(hipDoubleComplex* phases, 
				  hipDoubleComplex* deltaMinusBB,
				  hipDoubleComplex* deltaPlusBB,
				  hipDoubleComplex* twoEpsilonJBB,
				  int TLabLength) {

	int width = blockIdx.z * blockDim.z + threadIdx.z;
	hipDoubleComplex* twoEpsilonJ = new hipDoubleComplex[TLabLength];
	if (width < TLabLength) {
		twoEpsilonJ[width] = asinCudaComplex(sinCudaComplex(twoEpsilonJBB[width]) * sinCudaComplex(deltaMinusBB[width] - deltaPlusBB[width]));

		phases[0 + width*3] = 0.5 * (deltaPlusBB[width] + deltaMinusBB[width] + asinCudaComplex(tanCudaComplex(twoEpsilonJ[width]) / tanCudaComplex(twoEpsilonJBB[width]))) * constants::rad2deg;
		phases[1 + width*3] = 0.5 * (deltaPlusBB[width] + deltaMinusBB[width] - asinCudaComplex(tanCudaComplex(twoEpsilonJ[width]) / tanCudaComplex(twoEpsilonJBB[width]))) * constants::rad2deg;
		phases[2 + width*3] = 0.5 * twoEpsilonJ[width] * constants::rad2deg;
	}
}


/**
	Computes the phase shift for a given channel and T matrix.

	@param channel: Scattering channel
	@param key:		Channel name
	@param k0:		On-shell-point
	@param T:		T matrix
	@return			Complex phase shifts
*/
__global__
void computePhaseShifts(hipDoubleComplex* phases,
	hipDoubleComplex* T,
	double* k0,
	int quadratureN,
	double mu,
	bool coupled,
	int TLabLength,
	int matLength) {

	int width = blockIdx.z * blockDim.z + threadIdx.z;

	double* rhoT = new double[TLabLength];
	hipDoubleComplex* T11 = new hipDoubleComplex[TLabLength];
	hipDoubleComplex* T12 = new hipDoubleComplex[TLabLength];
	hipDoubleComplex* T22 = new hipDoubleComplex[TLabLength];
	hipDoubleComplex* twoEpsilonJBB = new hipDoubleComplex[TLabLength];
	hipDoubleComplex* deltaPlusBB = new hipDoubleComplex[TLabLength];
	hipDoubleComplex* deltaMinusBB = new hipDoubleComplex[TLabLength];

	if (width < TLabLength) {

		rhoT[width] = 2 * mu * k0[width]; // Equation (2.27) in the theory
		const hipDoubleComplex I = make_hipDoubleComplex(0.0, 1.0);

		// TODO: Explain theory for the phase shift for the coupled state
		if (coupled) {
			T11[width] = T[(quadratureN)+(quadratureN * quadratureN) + width * matLength * matLength]; //row + column * size
			T12[width] = T[(2 * quadratureN + 1) + (quadratureN * quadratureN) + width * matLength * matLength];
			T22[width] = T[(2 * quadratureN + 1) + (quadratureN * (2 * quadratureN + 1)) + width * matLength * matLength];

			//Blatt - Biedenharn(BB) convention
			twoEpsilonJBB[width] = atanCudaComplex(hipCdiv(hipCmul(make_hipDoubleComplex(2.0, 0), T12[width]), hipCsub(T11[width], T22[width])));
			deltaPlusBB[width] = -0.5 * I * logCudaComplex(1.0 - I * rhoT[width] * (T11[width] + T22[width]) + I * rhoT[width] * (2.0 * T12[width]) / sinCudaComplex(twoEpsilonJBB[width]));
			deltaMinusBB[width] = -0.5 * I * logCudaComplex(1.0 - I * rhoT[width] * (T11[width] + T22[width]) - I * rhoT[width] * (2.0 * T12[width]) / sinCudaComplex(twoEpsilonJBB[width]));

		}
		/* The uncoupled case completely follows equation (2.26). */
		else {
			double T0 = hipCreal(T[(quadratureN)+(quadratureN * quadratureN) + width * matLength ]); //Farligt, detta element kanske inte �r helt reellt. Dock var koden d�lig f�rut is�fall.
			hipDoubleComplex argument = make_hipDoubleComplex(1, -2.0 * rhoT[width] * T0);
			hipDoubleComplex swappedLog = make_hipDoubleComplex(hipCimag(logCudaComplex(argument)), hipCreal(logCudaComplex(argument)));
			hipDoubleComplex delta = hipCmul(make_hipDoubleComplex(-0.5 * constants::rad2deg, 0), swappedLog);
			phases[0] = delta;
		}
	}

	if (coupled) {
		blattToStapp(phases, deltaMinusBB, deltaPlusBB, twoEpsilonJBB, TLabLength);
	}
}