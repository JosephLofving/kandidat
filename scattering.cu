#include "hip/hip_runtime.h"
#include "scattering.h"


/**
	Sets up a complex vector needed to solve the T matrix equation.
	@param k:	Quadrature points
	@param w:	Weights for quadrature points
	@param k0:	On-shell-point
	@return		G0 vector
*/

hipDoubleComplex operator+(hipDoubleComplex A, hipDoubleComplex B) {
	hipDoubleComplex result = make_hipDoubleComplex(hipCreal(A)+hipCreal(B), hipCimag(A)+hipCimag(B));
	return result;
}

hipDoubleComplex operator-(hipDoubleComplex A, hipDoubleComplex B) {
	hipDoubleComplex result = make_hipDoubleComplex(hipCreal(A) - hipCreal(B), hipCimag(A) - hipCimag(B));
	return result;
}

hipDoubleComplex operator-(double a, hipDoubleComplex A) {
	hipDoubleComplex result = hipCsub(make_hipDoubleComplex(a, 0), A);
	return result;
}

hipDoubleComplex operator-(hipDoubleComplex A, double a) {
	hipDoubleComplex result = hipCsub(A, make_hipDoubleComplex(a, 0));
	return result;
}

hipDoubleComplex operator*(double scalar, hipDoubleComplex A) {
	hipDoubleComplex result = make_hipDoubleComplex(scalar * hipCreal(A), scalar * hipCimag(A));
	return result;
}

hipDoubleComplex operator*(hipDoubleComplex A, double scalar) {
	return scalar * A;
}

hipDoubleComplex operator*(hipDoubleComplex A, hipDoubleComplex B) {
	hipDoubleComplex realProd = hipCreal(A) * B;
	hipDoubleComplex imagProd = hipCimag(A) * B;
	hipDoubleComplex result = hipCadd(realProd, imagProd);
	return result;
}

hipDoubleComplex operator/(hipDoubleComplex A, hipDoubleComplex B) {
	return hipCdiv(A, B);
}

hipDoubleComplex operator/(hipDoubleComplex A, double a) {
	return hipCdiv(A, make_hipDoubleComplex(a, 0));
}

hipDoubleComplex operator/(double a, hipDoubleComplex A) {
	return hipCdiv(make_hipDoubleComplex(a, 0), A);
}

__device__
hipDoubleComplex logCudaComplex(hipDoubleComplex argument) {
	double x = hipCreal(argument);
	double y = hipCimag(argument);
	double real = logf(sqrtf(x * x + y * y));
	double imag = atan2f(y, x);
	hipDoubleComplex result = make_hipDoubleComplex(real, imag);
	return result;
}

__device__
double signCuda(double argument) {
	if (argument > 0) return 1;
	else if (argument == 0) return 0;
	else return -1;
}

__device__
hipDoubleComplex sqrtCudaComplex(hipDoubleComplex argument) {
	double x = hipCreal(argument);
	double y = hipCimag(argument);
	double real = sqrtf((sqrtf(x * x + y * y) + x) / 2);
	double imag = signCuda(y) * sqrtf((sqrtf(x * x + y * y) - x) / 2);
	return make_hipDoubleComplex(real, imag);
}

__device__
hipDoubleComplex atanCudaComplex(hipDoubleComplex argument) {
	hipDoubleComplex numerator = hipCadd(make_hipDoubleComplex(1, 0), hipCmul(make_hipDoubleComplex(0, 1), argument));
	hipDoubleComplex denominator = hipCsub(make_hipDoubleComplex(1, 0), hipCmul(make_hipDoubleComplex(0, 1), argument));
	hipDoubleComplex logOfStuff = logCudaComplex(hipCdiv(numerator, denominator));
	hipDoubleComplex result = hipCmul(make_hipDoubleComplex(0, -0.5), logOfStuff);
	return result;
}

__device__
hipDoubleComplex asinCudaComplex(hipDoubleComplex argument) {
	return I * logCudaComplex(sqrtCudaComplex(1 - argument * argument) - I * argument);
}

__device__
hipDoubleComplex sinCudaComplex(hipDoubleComplex argument) {
	return (expCudaComplex(I * argument) - expCudaComplex(-1.0 * I * argument)) / 2;
}

__device__
hipDoubleComplex tanCudaComplex(hipDoubleComplex argument) {
	hipDoubleComplex numerator = I * (expCudaComplex(-1.0 * I * argument) - expCudaComplex(I * argument));
	hipDoubleComplex denominator = expCudaComplex(-1.0 * I * argument) + expCudaComplex(I * argument);
	return numerator / denominator;
}

__device__
hipDoubleComplex expCudaComplex(hipDoubleComplex argument) {
	double x = hipCreal(argument);
	double y = hipCimag(argument);
	hipDoubleComplex trig = make_hipDoubleComplex(cosf(y), sinf(y));
	hipDoubleComplex result = make_hipDoubleComplex(expf(x), 0) * trig;
	return result;
}


__device__
void setupG0Vector(hipDoubleComplex* G0,
	double* k,
	double* w,
	double k0,
	int quadratureN,
	double mu,
	bool coupled) {

	double twoMu = (2.0 * mu);
	double twoOverPi = (2.0 / constants::pi);
	double sum = 0;
	for (int i = 0; i < quadratureN; i++) {
		G0[i] = make_hipDoubleComplex(twoOverPi * twoMu * k[i] * k[i] * w[i] / (k0 * k0 - k[i] * k[i]), 0);
		sum += w[i] / (k0 * k0 - k[i] * k[i]);

		/* If coupled, append G0 to itself to facilitate calculations.
		 * This means the second half of G0 is a copy of the first. */
		if (coupled) {
			G0[quadratureN + 1 + i] = G0[i];
		}
	}

	/* Assign the last element of D */
	G0[quadratureN] = make_hipDoubleComplex(-twoOverPi * twoMu * k0 * k0 * sum, -twoMu * k0);
	if (coupled) {
		G0[2 * (quadratureN + 1) - 1] = G0[quadratureN];
	}
}

/**
	Multiplies the potential matrix elements with the G0 vector elements.

	@param channel: Scattering channel
	@param key:		Channel name
	@param V:		Potential matrix
	@param k:		Quadrature points
	@param w:		Weights for quadrature points
	@param k0:		The on-shell-point
	@return			VG kernel
*/
__device__
void setupVGKernel(hipDoubleComplex* VG,
	hipDoubleComplex* V,
	hipDoubleComplex* G0,
	hipDoubleComplex* F,
	double* k,
	double* w,
	double k0,
	int quadratureN,
	int matSize,
	double mu,
	bool coupled) {

	setupG0Vector(G0, k, w, k0, quadratureN, mu, coupled);
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int column = blockIdx.x * blockDim.x + threadIdx.x;

	if (row < matSize && column < matSize) {
		VG[row + column * matSize] = hipCmul(V[row + column * matSize], G0[column]);
		//hipDoubleComplex test = hipCmul(V[row + column * matSize], G0[column]);
		hipDoubleComplex testG0 = G0[column];
		hipDoubleComplex testV = V[row + column * matSize];
		//printf("\nV = %f, %f", hipCreal(testV), hipCimag(testV));
		if (row == column) {
			F[row + row * matSize] = hipCadd(make_hipDoubleComplex(1, 0), hipCmul(make_hipDoubleComplex(-1, 0), VG[row + row * matSize])); // Diagonal element
		}
		else {
			F[row + column * matSize] = hipCmul(make_hipDoubleComplex(-1, 0), VG[row + column * matSize]);
		}

	}

}



	//for (int row = 0; row < matSize; row++) {
	//	for (int column = 0; column < matSize; column++) {
	//		/* Create VG by using VG[i,j] = V[i,j] * G[j] */
	//		VG[row + column * matSize] = hipCmul(V[row + column * matSize], G0[column]);

	//		/* At the same time, create F = delta_ij - VG_ij for computeTMatrix*/
	//		if (row != column) {
	//			F[row + column * matSize] = hipCmul(make_hipDoubleComplex(-1, 0), VG[row + column * matSize]);
	//		}
	//	}
	//	F[row + row * matSize] = hipCadd(make_hipDoubleComplex(1, 0), hipCmul(make_hipDoubleComplex(-1, 0), VG[row + row * matSize])); // Diagonal element
	//}
//}




/**
	Computes the T-matrix from the equation [F][T] = [V]

	@param channel: Scattering channel
	@param key:		Channel name
	@param V:		Potential matrix
	@param k:		Quadrature points
	@param w:		Weights for quadrature points
	@param k0:		On-shell-point
	@return			T matrix
*/
__global__
void computeTMatrix(hipDoubleComplex** T,
	hipDoubleComplex** V,
	hipDoubleComplex** G0,
	hipDoubleComplex** VG,
	hipDoubleComplex** F,
	double* k,
	double* w,
	double* k0,
	int quadratureN,
	int matSize,
	int TLabLength,
	double mu,
	bool coupled) {

	/* Setup the VG kernel and, at the same time, the F matrix */
	for (int i = 0; i < TLabLength; i++) {
		setupVGKernel(VG[i], V[i], G0[i], F[i], k, w, k0[i], quadratureN, matSize, mu, coupled);
	}

	/* Solve the equation FT = V with cuBLAS */
	//T = solveMatrixEq(F, V); // old lapack function
	// cuBLAS function here, hopefully takes in a parameter hipDoubleComplex pointer T and changes it
}



/* TODO: Explain theory for this. */
__device__
void blattToStapp(hipDoubleComplex* phases, hipDoubleComplex deltaMinusBB, hipDoubleComplex deltaPlusBB, hipDoubleComplex twoEpsilonJBB) {
	hipDoubleComplex twoEpsilonJ = asinCudaComplex(sinCudaComplex(twoEpsilonJBB) * sinCudaComplex(deltaMinusBB - deltaPlusBB));

	phases[0] = 0.5 * (deltaPlusBB + deltaMinusBB + asinCudaComplex(tanCudaComplex(twoEpsilonJ) / tanCudaComplex(twoEpsilonJBB))) * constants::rad2deg;
	phases[1] = 0.5 * (deltaPlusBB + deltaMinusBB - asinCudaComplex(tanCudaComplex(twoEpsilonJ) / tanCudaComplex(twoEpsilonJBB))) * constants::rad2deg;
	phases[2] = 0.5 * twoEpsilonJ * constants::rad2deg;
}


/**
	Computes the phase shift for a given channel and T matrix.

	@param channel: Scattering channel
	@param key:		Channel name
	@param k0:		On-shell-point
	@param T:		T matrix
	@return			Complex phase shifts
*/
__global__
void computePhaseShifts(hipDoubleComplex* phases, 
					    hipDoubleComplex* T, 
						double* k0, 
						int quadratureN, 
						double mu, 
						bool coupled) {
	
	double rhoT =  2 * mu * k0[0]; // Equation (2.27) in the theory

	// TODO: Explain theory for the phase shift for the coupled state
	if (coupled) {
		int N = quadratureN;
		hipDoubleComplex T11 = T[(N) + (N * N)]; //row + column * size
		hipDoubleComplex T12 = T[(2 * N + 1) + (N * N)];
		hipDoubleComplex T22 = T[(2 * N + 1) + (N * (2 * N + 1))];

		//Blatt - Biedenharn(BB) convention
		hipDoubleComplex twoEpsilonJBB = atanCudaComplex(hipCdiv(hipCmul(make_hipDoubleComplex(2.0, 0), T12), hipCsub(T11, T22)));
		hipDoubleComplex deltaPlusBB{ -0.5 * I * logCudaComplex(1.0 - I * rhoT * (T11 + T22) + I * rhoT * (2.0 * T12) / sinCudaComplex(twoEpsilonJBB)) };
		hipDoubleComplex deltaMinusBB{ -0.5 * I * logCudaComplex(1.0 - I * rhoT * (T11 + T22) - I * rhoT * (2.0 * T12) / sinCudaComplex(twoEpsilonJBB)) };
		blattToStapp(phases, deltaMinusBB, deltaPlusBB, twoEpsilonJBB);

	}
	/* The uncoupled case completely follows equation (2.26). */
	else {
		double T0 = hipCreal(T[(quadratureN) + (quadratureN * quadratureN)]); //Farligt, detta element kanske inte �r helt reellt. Dock var koden d�lig f�rut is�fall.
		hipDoubleComplex argument = make_hipDoubleComplex(1, -2.0 * rhoT * T0);
		hipDoubleComplex swappedLog = make_hipDoubleComplex(hipCimag(logCudaComplex(argument)), hipCreal(logCudaComplex(argument)));
		hipDoubleComplex delta = hipCmul(make_hipDoubleComplex(-0.5 * constants::rad2deg, 0), swappedLog);
		phases[0] = delta;
	}
}