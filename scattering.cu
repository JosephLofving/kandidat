#include "hip/hip_runtime.h"
#include "scattering.h"


/**
	Gets the reduced mass by checking the isospin channel, which determines the type of NN scattering
	@param channel:	Scattering channel
	@return			Reduced mass
*/
double getReducedMass(std::vector<QuantumState> channel) {
	double mu = 0;
	int tzChannel = channel[0].state["tz"];
	if (tzChannel == -1)	 // Proton-proton scattering
		mu = constants::protonMass / 2;
	else if (tzChannel == 0) // Proton-neutron scattering
		mu = constants::nucleonReducedMass;
	else if (tzChannel == 1) // Neutron-neutron scattering
		mu = constants::neutronMass / 2;

	return mu;
}


/** 
	Checks if the state is coupled or not. 
	@param channel: Scattering channel
	@return			True if coupled, false if not
*/
bool isCoupled(std::vector<QuantumState> channel) {
	return !(channel.size() == 1); // If there is only one channel the state is uncoupled, otherwise there are four channels and the state is coupled.
}


/** 
	Sets up a complex vector needed to solve the T matrix equation. 
	@param k:	Quadrature points
	@param w:	Weights for quadrature points
	@param k0:	On-shell-point
	@return		G0 vector
*/

__device__
hipDoubleComplex* setupG0Vector(double mu, double* k, double* w, double k0, int Nkvadr) {
	hipDoubleComplex* D = new hipDoubleComplex[Nkvadr + 1];

	double twoMu = (2.0 * mu);
	double twoOverPi = (2.0 / constants::pi);
	double sum = 0;


	for (int i = 0; i < Nkvadr; i++) {
		D[i] = make_hipDoubleComplex(-twoOverPi * twoMu * pow(k[i], 2) * w[i] / (pow(k0, 2) - pow(k[i], 2)), 0); // Define D[0,N-1] with vectors k and w
		sum += w[i] / (pow(k0, 2) - pow(k[i], 2));																// Used in D[N]
	}

	D[Nkvadr] = make_hipDoubleComplex(twoOverPi * twoMu * pow(k0, 2) * sum, twoMu * k0);	// In the theory, this element is placed at index 0

	return D;
}

/**
	Multiplies the potential matrix with the G0 vector.

	@param channel: Scattering channel
    @param key:		Channel name
    @param V:		Potential matrix
	@param k:		Quadrature points
	@param w:		Weights for quadrature points
	@param k0:		The on-shell-point
	@return			VG kernel
*/

__global__
void setupVGKernel(hipDoubleComplex* VG, double mu, bool coupled, hipDoubleComplex* V, double* k, double* w, double k0, int Nkvadr, int G0Size) {
	
	hipDoubleComplex* G0 = setupG0Vector(mu, k, w, k0, Nkvadr);

	/* If coupled, append G0 to itself to facilitate calculations. This means the second half of G0 is a copy of the first. */
	if (coupled) {
		hipDoubleComplex* G1 = new hipDoubleComplex[2*(Nkvadr + 1)];
		for (int i = 0; i < Nkvadr + 1; ++i) {
			G1[i] = G0[i];
			G1[Nkvadr + 1 + i] = G0[i];
		}
		hipDoubleComplex* G0 = G1;

	}

	/* Create VG by using VG[i,j] = V[i,j] * G[j] */
	hipDoubleComplex* VElement = new hipDoubleComplex[1];
	for (int row = 0; row < G0Size; row++) {
		for (int column = 0; column < G0Size; column++) {
			//VG[row + column * G0Size] = make_hipDoubleComplex(hipCreal(hipCmul(V[row + column * G0Size], G0[column])),
			//												 hipCimag(hipCmul(V[row + column * G0Size], G0[column])));
			VG[row + column * G0Size] = hipCmul(V[row + column * G0Size], G0[column]);
		}
	}

	//for (int i = 0; i < G0Size * G0Size; i += 100) {
	//	std::cout << VG.contents[i].real() << std::endl;
	//}

	//funktionen returnar VG
}


/**
	Computes the T-matrix from the equation [F][T] = [V]

	@param channel: Scattering channel
	@param key:		Channel name
	@param V:		Potential matrix
	@param k:		Quadrature points
	@param w:		Weights for quadrature points
	@param k0:		On-shell-point
	@return			T matrix
*/
hipDoubleComplex* computeTMatrix(LapackMat V_matrix, double* k, double* w, double k0, int Nkvadr, int G0Size, double mu, bool coupled)  {

	hipDoubleComplex* V_host = new hipDoubleComplex[V_matrix.width * V_matrix.height];
	for (int i = 0; i < Nkvadr * Nkvadr; i++) {
		V_host[i] = make_hipDoubleComplex(V_matrix.contents[i].real(), V_matrix.contents[i].imag());
	}

	hipDoubleComplex* VG = new hipDoubleComplex[G0Size * G0Size];
	setupVGKernel(VG, mu, coupled, V_host, k, w, k0, Nkvadr, G0Size);

	hipDoubleComplex* F = new hipDoubleComplex[G0Size * G0Size];
	for (int i = 0; i < G0Size; ++i) {
		F[i + i * G0Size] = hipCadd(VG[i + i * G0Size], make_hipDoubleComplex(1, 0));
	}

	// Solves the equation FT = V.
	hipDoubleComplex* T = solveMatrixEq(F, V_host); //Josephs problem :)

	return T;
}


/* TODO: Explain theory for this. */
std::vector<std::complex<double>> blattToStapp(std::complex<double> deltaMinusBB, std::complex<double> deltaPlusBB, std::complex<double> twoEpsilonJBB)
{
	std::complex<double> twoEpsilonJ = std::asin(std::sin(twoEpsilonJBB) * std::sin(deltaMinusBB - deltaPlusBB));

	std::complex<double> deltaMinus = 0.5 * (deltaPlusBB + deltaMinusBB + std::asin(tan(twoEpsilonJ) / std::tan(twoEpsilonJBB))) * constants::rad2deg;
	std::complex<double> deltaPlus = 0.5 * (deltaPlusBB + deltaMinusBB - std::asin(tan(twoEpsilonJ) / std::tan(twoEpsilonJBB))) * constants::rad2deg;
	std::complex<double> epsilon = 0.5 * twoEpsilonJ * constants::rad2deg;

	return { deltaMinus, deltaPlus, epsilon };
}


/**
	Computes the phase shift for a given channel and T matrix.

	@param channel: Scattering channel
	@param key:		Channel name
	@param k0:		On-shell-point
	@param T:		T matrix
	@return			Complex phase shifts
*/

__global__
std::vector<std::complex<double>> computePhaseShifts(hipDoubleComplex* phases, double mu, bool coupled, std::string key, double k0, hipDoubleComplex* T, int Nkvadr) {
	
	double rhoT =  2 * mu * k0; // Equation (2.27) in the theory

	// TODO: Explain theory for the phase shift for the coupled state
	if (coupled) {
		/*int N = Nkvadr;
		hipDoubleComplex T11 = T[(N) + (N * N)]; //row + column * size
		hipDoubleComplex T12 = T[(2 * N + 1) + (N * N)];
		hipDoubleComplex T22 = T[(2 * N + 1) + (N * (2 * N + 1))];

		//Blatt - Biedenharn(BB) convention
		std::complex<double> twoEpsilonJBB{ std::atan(2.0 * T12 / (T11 - T22)) };
		std::complex<double> deltaPlusBB{ -0.5 * I * std::log(1.0 - I * rhoT * (T11 + T22) + I * rhoT * (2.0 * T12) / std::sin(twoEpsilonJBB)) };
		std::complex<double> deltaMinusBB{ -0.5 * I * std::log(1.0 - I * rhoT * (T11 + T22) - I * rhoT * (2.0 * T12) / std::sin(twoEpsilonJBB)) };

		std::vector<std::complex<double>> phasesAppend{ blattToStapp(deltaMinusBB, deltaPlusBB, twoEpsilonJBB) };

		phases.push_back(phasesAppend[0]);
		phases.push_back(phasesAppend[1]);
		phases.push_back(phasesAppend[2]); 

		*/
		//avkommenterade f�r de ger error vid icke kopplad kompilering. Avkommentera och fixa.
	}
	/* The uncoupled case completely follows equation (2.26). */
	else {
		double T0 = hipCreal(T[(Nkvadr) + (Nkvadr * Nkvadr)]); //Farligt, detta element kanske inte �r helt reelt. Dock var koden d�lig f�rut is�fall.
		hipDoubleComplex argument = make_hipDoubleComplex(1, -2.0 * rhoT * T0);
		hipDoubleComplex delta = (-0.5 * I) * logf(argument) * constants::rad2deg;

		phases.push_back(delta);
	}

	return phases;
}