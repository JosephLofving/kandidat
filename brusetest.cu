#include "hip/hip_runtime.h"
#include "mesh.h"
#include "scattering.h"
#include "potential.h"
#include <fstream>
#include <iomanip>
#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

__global__
void setupVG(double* k)
{	
	//int matrixHeight = 100;
	k[0] *= 100;
}


int main() {
	int N = 100;
	TwoVectors k_and_w = gaussLegendreInfMesh(100, 100);

	std::vector<double> kVect = k_and_w.v1;
	double* k = &kVect[0];
	double* k_dev;
	hipMalloc((void**)&k_dev, N * sizeof(double));
	hipMemcpy(&k_dev, &k, N * sizeof(double), hipMemcpyHostToDevice);

	std::cout << k_dev[0];
	setupVG << <1, 1 >> > (k_dev);

	hipDeviceSynchronize();

	std::cout << k_dev[0];
	hipFree(k_dev);
	std::cout << "hej";

	return 0;
}