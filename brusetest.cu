#include "hip/hip_runtime.h"
#include "mesh.h"
#include "scattering.h"
#include "potential.h"
#include <fstream>
#include <iomanip>
#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

__global__
void setupVG(double* a)
{	
	a[0] *= 100;
}


int main() {
	int N = 100;
	TwoVectors k_and_w = gaussLegendreInfMesh(100, 100);

	std::vector<double> kVect = k_and_w.v1;
	double* k = &kVect[0];
	double* k_dev;
	hipMalloc((void**)&k_dev, N * sizeof(double));
	hipMemcpy(&k_dev, &k, N * sizeof(double), hipMemcpyHostToDevice);

	std::cout << k[0] << std::endl;
	setupVG << <1, 1 >> > (k_dev);
	hipMemcpy(&k, &k_dev, N * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);

	hipDeviceSynchronize();

	std::cout << k[0] << std::endl;
	hipFree(k_dev);
	std::cout << "hej" << std::endl;

	return 0;
}